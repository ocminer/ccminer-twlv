#include "hip/hip_runtime.h"
#include <memory.h>

#include "cuda_helper.h"

#if 0
static __constant__ uint64_t SKEIN_IV512_256[8] = {
	0xCCD044A12FDB3E13, 0xE83590301A79A9EB,
	0x55AEA0614F816E6F, 0x2A2767A4AE9B94DB,
	0xEC06025E74DD7683, 0xE7A436CDC4746251,
	0xC36FBAF9393AD185, 0x3EEDBA1833EDFC13
};
#endif

static __constant__ uint2 vSKEIN_IV512_256[8] = {
	{ 0x2FDB3E13, 0xCCD044A1 },
	{ 0x1A79A9EB, 0xE8359030 },
	{ 0x4F816E6F, 0x55AEA061 },
	{ 0xAE9B94DB, 0x2A2767A4 },
	{ 0x74DD7683, 0xEC06025E },
	{ 0xC4746251, 0xE7A436CD },
	{ 0x393AD185, 0xC36FBAF9 },
	{ 0x33EDFC13, 0x3EEDBA18 }
};

static __constant__ int ROT256[8][4] =
{
	46,36, 19, 37,
	33,27, 14, 42,
	17,49, 36, 39,
	44, 9, 54, 56,
	39,30, 34, 24,
	13,50, 10, 17,
	25,29, 39, 43,
	8, 35, 56, 22,
};

static __constant__ uint2 skein_ks_parity = { 0xA9FC1A22,0x1BD11BDA};
static __constant__ uint2 t12[6] = {
	{ 0x20,	0 },
	{ 0,	0xf0000000 },
	{ 0x20,	0xf0000000 },
	{ 0x08,	0 },
	{ 0,	0xff000000 },
	{ 0x08,	0xff000000 }
};

#if 0
static __constant__ uint64_t t12_30[6] = {
	0x20,
	0xf000000000000000,
	0xf000000000000020,
	0x08,
	0xff00000000000000,
	0xff00000000000008
};
#endif

static __forceinline__ __device__
void Round512v35(uint2 &p0, uint2 &p1, uint2 &p2, uint2 &p3, uint2 &p4, uint2 &p5, uint2 &p6, uint2 &p7, int ROT)
{
	p0 += p1; p1 = ROL2(p1, ROT256[ROT][0]);  p1 ^= p0;
	p2 += p3; p3 = ROL2(p3, ROT256[ROT][1]);  p3 ^= p2;
	p4 += p5; p5 = ROL2(p5, ROT256[ROT][2]);  p5 ^= p4;
	p6 += p7; p7 = ROL2(p7, ROT256[ROT][3]);  p7 ^= p6;
}


static __forceinline__ __device__
void Round_8_512v35(uint2 *ks, uint2 *ts,
                    uint2 &p0, uint2 &p1, uint2 &p2, uint2 &p3,
                    uint2 &p4, uint2 &p5, uint2 &p6, uint2 &p7, int R)
{
	Round512v35(p0, p1, p2, p3, p4, p5, p6, p7, 0);
	Round512v35(p2, p1, p4, p7, p6, p5, p0, p3, 1);
	Round512v35(p4, p1, p6, p3, p0, p5, p2, p7, 2);
	Round512v35(p6, p1, p0, p7, p2, p5, p4, p3, 3);
	p0 += ks[((R)+0) % 9];   /* inject the key schedule value */
	p1 += ks[((R)+1) % 9];
	p2 += ks[((R)+2) % 9];
	p3 += ks[((R)+3) % 9];
	p4 += ks[((R)+4) % 9];
	p5 += ks[((R)+5) % 9] + ts[((R)+0) % 3];
	p6 += ks[((R)+6) % 9] + ts[((R)+1) % 3];
	p7 += ks[((R)+7) % 9] + make_uint2((R),0);
	Round512v35(p0, p1, p2, p3, p4, p5, p6, p7, 4);
	Round512v35(p2, p1, p4, p7, p6, p5, p0, p3, 5);
	Round512v35(p4, p1, p6, p3, p0, p5, p2, p7, 6);
	Round512v35(p6, p1, p0, p7, p2, p5, p4, p3, 7);
	p0 += ks[((R)+1) % 9];   /* inject the key schedule value */
	p1 += ks[((R)+2) % 9];
	p2 += ks[((R)+3) % 9];
	p3 += ks[((R)+4) % 9];
	p4 += ks[((R)+5) % 9];
	p5 += ks[((R)+6) % 9] + ts[((R)+1) % 3];
	p6 += ks[((R)+7) % 9] + ts[((R)+2) % 3];
	p7 += ks[((R)+8) % 9] + make_uint2((R)+1, 0);
}


__global__ __launch_bounds__(256,3)
void skein256_gpu_hash_32(uint32_t threads, uint32_t startNounce, uint64_t *outputHash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint2 h[9];
		uint2 t[3];
		uint2 dt0,dt1,dt2,dt3;
		uint2 p0, p1, p2, p3, p4, p5, p6, p7;

		h[8] = skein_ks_parity;
		for (int i = 0; i<8; i++) {
			h[i] = vSKEIN_IV512_256[i];
			h[8] ^= h[i];
		}

		t[0]=t12[0];
		t[1]=t12[1];
		t[2]=t12[2];

		LOHI(dt0.x,dt0.y,outputHash[thread]);
		LOHI(dt1.x,dt1.y,outputHash[threads+thread]);
		LOHI(dt2.x,dt2.y,outputHash[2*threads+thread]);
		LOHI(dt3.x,dt3.y,outputHash[3*threads+thread]);

		p0 = h[0] + dt0;
		p1 = h[1] + dt1;
		p2 = h[2] + dt2;
		p3 = h[3] + dt3;
		p4 = h[4];
		p5 = h[5] + t[0];
		p6 = h[6] + t[1];
		p7 = h[7];

		#pragma unroll
		for (int i = 1; i<19; i+=2) {
			Round_8_512v35(h,t,p0,p1,p2,p3,p4,p5,p6,p7,i);
		}

		p0 ^= dt0;
		p1 ^= dt1;
		p2 ^= dt2;
		p3 ^= dt3;

		h[0] = p0;
		h[1] = p1;
		h[2] = p2;
		h[3] = p3;
		h[4] = p4;
		h[5] = p5;
		h[6] = p6;
		h[7] = p7;
		h[8] = skein_ks_parity;

		#pragma unroll 8
		for (int i = 0; i<8; i++) {
			h[8] ^= h[i];
		}

		t[0] = t12[3];
		t[1] = t12[4];
		t[2] = t12[5];
		p5 += t[0];  //p5 already equal h[5]
		p6 += t[1];

		#pragma unroll
		for (int i = 1; i<19; i+=2) {
			Round_8_512v35(h, t, p0, p1, p2, p3, p4, p5, p6, p7, i);
		}

		outputHash[thread]           = devectorize(p0);
		outputHash[threads+thread]   = devectorize(p1);
		outputHash[2*threads+thread] = devectorize(p2);
		outputHash[3*threads+thread] = devectorize(p3);
	}
}

__host__
void skein256_cpu_init(int thr_id, uint32_t threads)
{
	//empty
}

__host__
void skein256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_outputHash, int order)
{
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	skein256_gpu_hash_32<<<grid, block>>>(threads, startNounce, d_outputHash);

}

