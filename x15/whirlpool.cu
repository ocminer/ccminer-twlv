#include "hip/hip_runtime.h"
/*
 * whirlpool routine (djm)
 */
extern "C"
{
#include "sph/sph_whirlpool.h"
#include "miner.h"
}

#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS];

extern void x15_whirlpool_cpu_init(int thr_id, uint32_t threads, int mode);
extern void x15_whirlpool_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void whirlpool512_setBlock_80(void *pdata, const void *ptarget);
extern void whirlpool512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order);
extern uint32_t whirlpool512_cpu_finalhash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);


// CPU Hash function
extern "C" void wcoinhash(void *state, const void *input)
{
	sph_whirlpool_context ctx_whirlpool;

	unsigned char hash[128]; // uint32_t hashA[16], hashB[16];
	#define hashB hash+64

	memset(hash, 0, sizeof hash);

	// shavite 1
	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, input, 80);
	sph_whirlpool1_close(&ctx_whirlpool, hash);

	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, hash, 64);
	sph_whirlpool1_close(&ctx_whirlpool, hashB);

	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, hashB, 64);
	sph_whirlpool1_close(&ctx_whirlpool, hash);

	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, hash, 64);
	sph_whirlpool1_close(&ctx_whirlpool, hash);

	memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_whc(int thr_id, uint32_t *pdata,
    const uint32_t *ptarget, uint32_t max_nonce,
    unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	uint32_t endiandata[20];
	uint32_t throughput = device_intensity(thr_id, __func__, 1U << 19); // 19=256*256*8;
	throughput = min(throughput, (max_nonce - first_nonce));

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0000ff;

	if (!init[thr_id]) {
		CUDA_CALL_OR_RET_X(hipSetDevice(device_map[thr_id]), 0);
		// Konstanten kopieren, Speicher belegen
		hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput);
		x15_whirlpool_cpu_init(thr_id, throughput, 1 /* old whirlpool */);

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++) {
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);
	}

	whirlpool512_setBlock_80((void*)endiandata, ptarget);

	do {
		uint32_t foundNonce;
		int order = 0;

		whirlpool512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		x15_whirlpool_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x15_whirlpool_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		foundNonce = whirlpool512_cpu_finalhash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		if (foundNonce != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);
			wcoinhash(vhash64, endiandata);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget)) {
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
				#if 0
				uint32_t secNonce = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				if (secNonce != 0) {
					pdata[21] = secNonce;
					res++;
				}
				#endif
				pdata[19] = foundNonce;
				return res;
			}
			else if (vhash64[7] > Htarg) {
				applog(LOG_INFO, "GPU #%d: result for %08x is not in range: %x > %x", thr_id, foundNonce, vhash64[7], Htarg);
			}
			else {
				applog(LOG_INFO, "GPU #%d: result for %08x does not validate on CPU!", thr_id, foundNonce);
			}
		}
		pdata[19] += throughput;
	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}
