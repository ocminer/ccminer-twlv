#include "hip/hip_runtime.h"
/*
 * qubit algorithm
 *
 */
extern "C" {
#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"
}

#include "miner.h"

#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS];
static uint32_t *h_found[MAX_GPUS];

extern void qubit_luffa512_cpu_init(int thr_id, uint32_t threads);
extern void qubit_luffa512_cpu_setBlock_80(void *pdata);
extern void qubit_luffa512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order);

extern void x11_cubehash512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x11_shavite512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern int x11_simd512_cpu_init(int thr_id, uint32_t threads);
extern void x11_simd512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x11_echo512_cpu_init(int thr_id, uint32_t threads);
extern void x11_echo512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
extern void x11_echo512_cpu_hash_64_final(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, uint32_t target, uint32_t *h_found, int order);

extern void quark_compactTest_cpu_init(int thr_id, uint32_t threads);
extern void quark_compactTest_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *inpHashes,
											uint32_t *d_noncesTrue, uint32_t *nrmTrue, uint32_t *d_noncesFalse, uint32_t *nrmFalse,
											int order);

extern "C" void qubithash(void *state, const void *input)
{
	// luffa1-cubehash2-shavite3-simd4-echo5

	sph_luffa512_context ctx_luffa;
	sph_cubehash512_context ctx_cubehash;
	sph_shavite512_context ctx_shavite;
	sph_simd512_context ctx_simd;
	sph_echo512_context ctx_echo;

	uint8_t hash[64];

	sph_luffa512_init(&ctx_luffa);
	sph_luffa512 (&ctx_luffa, input, 80);
	sph_luffa512_close(&ctx_luffa, (void*) hash);

	sph_cubehash512_init(&ctx_cubehash);
	sph_cubehash512 (&ctx_cubehash, (const void*) hash, 64);
	sph_cubehash512_close(&ctx_cubehash, (void*) hash);

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512 (&ctx_shavite, (const void*) hash, 64);
	sph_shavite512_close(&ctx_shavite, (void*) hash);

	sph_simd512_init(&ctx_simd);
	sph_simd512 (&ctx_simd, (const void*) hash, 64);
	sph_simd512_close(&ctx_simd, (void*) hash);

	sph_echo512_init(&ctx_echo);
	sph_echo512 (&ctx_echo, (const void*) hash, 64);
	sph_echo512_close(&ctx_echo, (void*) hash);

	memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_qubit(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	uint32_t endiandata[20];
	const uint32_t first_nonce = pdata[19];
	uint32_t throughput = device_intensity(thr_id, __func__, 1U << 19); // 256*256*8
	throughput = min(throughput, (max_nonce - first_nonce));

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0000ff;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		hipDeviceReset();
		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		hipDeviceSetCacheConfig(hipFuncCachePreferL1);

		qubit_luffa512_cpu_init(thr_id, throughput);
		x11_simd512_cpu_init(thr_id, throughput);
		x11_echo512_cpu_init(thr_id, throughput);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput), 0);
		CUDA_CALL_OR_RET_X(hipHostMalloc(&(h_found[thr_id]), 4 * sizeof(uint32_t)), 0);

		cuda_check_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);

	qubit_luffa512_cpu_setBlock_80((void*)endiandata);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;

		// Hash with CUDA
		qubit_luffa512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		x11_cubehash512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_echo512_cpu_hash_64_final(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], ptarget[7], h_found[thr_id], order++);
		if (h_found[thr_id][0] != 0xffffffff)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], h_found[thr_id][0]);
			qubithash(vhash64, endiandata);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				// check if there was some other ones...
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (h_found[thr_id][1] != 0xffffffff)
				{
					pdata[21] = h_found[thr_id][1];
					res++;
					if (opt_benchmark)
						applog(LOG_INFO, "GPU #%d Found second nounce %08x", thr_id, h_found[thr_id][1], vhash64[7], Htarg);
				}
				pdata[19] = h_found[thr_id][0];
				if (opt_benchmark)
					applog(LOG_INFO, "GPU #%d Found nounce %08x", thr_id, h_found[thr_id][0], vhash64[7], Htarg);
				return res;
			}
			else
			{
				if (vhash64[7] != Htarg)
				{
					applog(LOG_INFO, "GPU #%d: result for %08x does not validate on CPU!", thr_id, h_found[thr_id][0]);
				}
			}
		}

		pdata[19] += throughput;
	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
