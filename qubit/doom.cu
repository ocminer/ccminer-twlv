#include "hip/hip_runtime.h"
/*
 * qubit algorithm
 *
 */
extern "C" {
#include "sph/sph_luffa.h"
}

#include "miner.h"

#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS];

extern void qubit_luffa512_cpu_init(int thr_id, uint32_t threads);
extern void qubit_luffa512_cpu_setBlock_80(void *pdata);
extern void qubit_luffa512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order);
extern void qubit_luffa512_cpufinal_setBlock_80(void *pdata, const void *ptarget);
extern uint32_t qubit_luffa512_cpu_finalhash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order);

extern void doomhash(void *state, const void *input)
{
	// luffa512
	sph_luffa512_context ctx_luffa;

	uint8_t hash[64];

	sph_luffa512_init(&ctx_luffa);
	sph_luffa512 (&ctx_luffa, input, 80);
	sph_luffa512_close(&ctx_luffa, (void*) hash);

	memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_doom(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	uint32_t endiandata[20];
	uint32_t throughput = device_intensity(thr_id, __func__, 1U << 22); // 256*256*8*8
	throughput = min(throughput, (max_nonce - first_nonce));

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0000f;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		hipDeviceReset();
		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		hipDeviceSetCacheConfig(hipFuncCachePreferL1);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput));

		qubit_luffa512_cpu_init(thr_id, (int) throughput);

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);

	qubit_luffa512_cpufinal_setBlock_80((void*)endiandata,ptarget);

	do {
		int order = 0;

		uint32_t foundNonce = qubit_luffa512_cpu_finalhash_80(thr_id, (int) throughput, pdata[19], d_hash[thr_id], order++);
		if (foundNonce != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);
			doomhash(vhash64, endiandata);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget)) {
				*hashes_done = min(max_nonce - first_nonce, (uint64_t) pdata[19] - first_nonce + throughput);
				pdata[19] = foundNonce;
				return 1;
			}
			else {
				applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU!", thr_id, foundNonce);
			}
		}

		pdata[19] += throughput;
	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
