#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"

#define ROTR(x,n) ROTR64(x,n)

#define USE_SHUFFLE 0

// die Message it Padding zur Berechnung auf der GPU
__constant__ uint64_t c_PaddedMessage80[16]; // padded message (80 bytes + padding)

// ---------------------------- BEGIN CUDA quark_blake512 functions ------------------------------------


#define Gprecalc(a,b,c,d,idx1,idx2) { \
	v[a] += (block[idx2] ^ u512[idx1]) + v[b]; \
	v[d] = SWAPDWORDS2( v[d] ^ v[a]); \
	v[c] += v[d]; \
	v[b] = ROR2(v[b] ^ v[c], 25); \
	v[a] += (block[idx1] ^ u512[idx2]) + v[b]; \
	v[d] = ROR2(v[d] ^ v[a],16); \
	v[c] += v[d]; \
	v[b] = ROR2(v[b] ^ v[c], 11); \
	}

__global__ 
#if __CUDA_ARCH__ > 500
	__launch_bounds__(256, 1)
#else
	__launch_bounds__(256, 2)
#endif
void quark_blake512_gpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *const __restrict__ g_nonceVector, uint64_t *const __restrict__ g_hash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

#if USE_SHUFFLE
	const int warpID = threadIdx.x & 0x0F; // 16 warps
	const int warpBlockID = (thread + 15)>>4; // aufrunden auf volle Warp-Bl�cke
	const int maxHashPosition = thread<<3;
#endif

#if USE_SHUFFLE
	if (warpBlockID < ( (threads+15)>>4 ))
#else
	if (thread < threads)
#endif
	{
		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

		int hashPosition = nounce - startNounce;

		uint64_t *inpHash = &g_hash[hashPosition*8];
		uint2 block[16] =
		{
			vectorizeswap(inpHash[0]), vectorizeswap(inpHash[1]), vectorizeswap(inpHash[2]), vectorizeswap(inpHash[3]),
			vectorizeswap(inpHash[4]), vectorizeswap(inpHash[5]), vectorizeswap(inpHash[6]), vectorizeswap(inpHash[7])
		};
		block[8] = make_uint2(0, 0x80000000UL);
		block[9] = make_uint2(0,0);
		block[10] = make_uint2(0,0);
		block[11] = make_uint2(0,0);
		block[12] = make_uint2(0,0);
		block[13] = make_uint2(1,0);
		block[14] = make_uint2(0,0);
		block[15] = make_uint2(0x200,0);
		const uint2 h[8] =
		{
				{ 0xf3bcc908UL, 0x6a09e667UL },
				{ 0x84caa73bUL, 0xbb67ae85UL },
				{ 0xfe94f82bUL, 0x3c6ef372UL },
				{ 0x5f1d36f1UL, 0xa54ff53aUL },
				{ 0xade682d1UL, 0x510e527fUL },
				{ 0x2b3e6c1fUL, 0x9b05688cUL },
				{ 0xfb41bd6bUL, 0x1f83d9abUL },
				{ 0x137e2179UL, 0x5be0cd19UL }
		};
		const uint2 u512[16] =
		{
			{ 0x85a308d3UL, 0x243f6a88 }, { 0x03707344UL, 0x13198a2e },
			{ 0x299f31d0UL, 0xa4093822 }, { 0xec4e6c89UL, 0x082efa98 },
			{ 0x38d01377UL, 0x452821e6 }, { 0x34e90c6cUL, 0xbe5466cf },
			{ 0xc97c50ddUL, 0xc0ac29b7 }, { 0xb5470917UL, 0x3f84d5b5 },
			{ 0x8979fb1bUL, 0x9216d5d9 }, { 0x98dfb5acUL, 0xd1310ba6 },
			{ 0xd01adfb7UL, 0x2ffd72db }, { 0x6a267e96UL, 0xb8e1afed },
			{ 0xf12c7f99UL, 0xba7c9045 }, { 0xb3916cf7UL, 0x24a19947 },
			{ 0x858efc16UL, 0x0801f2e2 }, { 0x71574e69UL, 0x636920d8 }
		};

		uint2 v[16] =
		{
			h[0], h[1], h[2], h[3], h[4], h[5], h[6], h[7],
			u512[0], u512[1], u512[2], u512[3], u512[4] ^ 512, u512[5] ^ 512, u512[6], u512[7]
		};

		Gprecalc(0, 4, 8, 12, 0x1, 0x0)
		Gprecalc(1, 5, 9, 13, 0x3, 0x2)
		Gprecalc(2, 6, 10, 14, 0x5, 0x4)
		Gprecalc(3, 7, 11, 15, 0x7, 0x6)
		Gprecalc(0, 5, 10, 15, 0x9, 0x8)
		Gprecalc(1, 6, 11, 12, 0xb, 0xa)
		Gprecalc(2, 7, 8, 13, 0xd, 0xc)
		Gprecalc(3, 4, 9, 14, 0xf, 0xe)

		Gprecalc(0, 4, 8, 12, 0xa, 0xe)
		Gprecalc(1, 5, 9, 13, 0x8, 0x4)
		Gprecalc(2, 6, 10, 14, 0xf, 0x9)
		Gprecalc(3, 7, 11, 15, 0x6, 0xd)
		Gprecalc(0, 5, 10, 15, 0xc, 0x1)
		Gprecalc(1, 6, 11, 12, 0x2, 0x0)
		Gprecalc(2, 7, 8, 13, 0x7, 0xb)
		Gprecalc(3, 4, 9, 14, 0x3, 0x5)

		Gprecalc(0, 4, 8, 12, 0x8, 0xb)
		Gprecalc(1, 5, 9, 13, 0x0, 0xc)
		Gprecalc(2, 6, 10, 14, 0x2, 0x5)
		Gprecalc(3, 7, 11, 15, 0xd, 0xf)
		Gprecalc(0, 5, 10, 15, 0xe, 0xa)
		Gprecalc(1, 6, 11, 12, 0x6, 0x3)
		Gprecalc(2, 7, 8, 13, 0x1, 0x7)
		Gprecalc(3, 4, 9, 14, 0x4, 0x9)
		
		Gprecalc(0, 4, 8, 12, 0x9, 0x7)
		Gprecalc(1, 5, 9, 13, 0x1, 0x3)
		Gprecalc(2, 6, 10, 14, 0xc, 0xd)
		Gprecalc(3, 7, 11, 15, 0xe, 0xb)
		Gprecalc(0, 5, 10, 15, 0x6, 0x2)
		Gprecalc(1, 6, 11, 12, 0xa, 0x5)
		Gprecalc(2, 7, 8, 13, 0x0, 0x4)
		Gprecalc(3, 4, 9, 14, 0x8, 0xf)
		
		Gprecalc(0, 4, 8, 12, 0x0, 0x9)
		Gprecalc(1, 5, 9, 13, 0x7, 0x5)
		Gprecalc(2, 6, 10, 14, 0x4, 0x2)
		Gprecalc(3, 7, 11, 15, 0xf, 0xa)
		Gprecalc(0, 5, 10, 15, 0x1, 0xe)
		Gprecalc(1, 6, 11, 12, 0xc, 0xb)
		Gprecalc(2, 7, 8, 13, 0x8, 0x6)
		Gprecalc(3, 4, 9, 14, 0xd, 0x3)
		
		Gprecalc(0, 4, 8, 12, 0xc, 0x2)
		Gprecalc(1, 5, 9, 13, 0xa, 0x6)
		Gprecalc(2, 6, 10, 14, 0xb, 0x0)
		Gprecalc(3, 7, 11, 15, 0x3, 0x8)
		Gprecalc(0, 5, 10, 15, 0xd, 0x4)
		Gprecalc(1, 6, 11, 12, 0x5, 0x7)
		Gprecalc(2, 7, 8, 13, 0xe, 0xf)
		Gprecalc(3, 4, 9, 14, 0x9, 0x1)
		
		Gprecalc(0, 4, 8, 12, 0x5, 0xc)
		Gprecalc(1, 5, 9, 13, 0xf, 0x1)
		Gprecalc(2, 6, 10, 14, 0xd, 0xe)
		Gprecalc(3, 7, 11, 15, 0xa, 0x4)
		Gprecalc(0, 5, 10, 15, 0x7, 0x0)
		Gprecalc(1, 6, 11, 12, 0x3, 0x6)
		Gprecalc(2, 7, 8, 13, 0x2, 0x9)
		Gprecalc(3, 4, 9, 14, 0xb, 0x8)
		
		Gprecalc(0, 4, 8, 12, 0xb, 0xd)
		Gprecalc(1, 5, 9, 13, 0xe, 0x7)
		Gprecalc(2, 6, 10, 14, 0x1, 0xc)
		Gprecalc(3, 7, 11, 15, 0x9, 0x3)
		Gprecalc(0, 5, 10, 15, 0x0, 0x5)
		Gprecalc(1, 6, 11, 12, 0x4, 0xf)
		Gprecalc(2, 7, 8, 13, 0x6, 0x8)
		Gprecalc(3, 4, 9, 14, 0xa, 0x2)
		
		Gprecalc(0, 4, 8, 12, 0xf, 0x6)
		Gprecalc(1, 5, 9, 13, 0x9, 0xe)
		Gprecalc(2, 6, 10, 14, 0x3, 0xb)
		Gprecalc(3, 7, 11, 15, 0x8, 0x0)
		Gprecalc(0, 5, 10, 15, 0x2, 0xc)
		Gprecalc(1, 6, 11, 12, 0x7, 0xd)
		Gprecalc(2, 7, 8, 13, 0x4, 0x1)
		Gprecalc(3, 4, 9, 14, 0x5, 0xa)
		
		Gprecalc(0, 4, 8, 12, 0x2, 0xa)
		Gprecalc(1, 5, 9, 13, 0x4, 0x8)
		Gprecalc(2, 6, 10, 14, 0x6, 0x7)
		Gprecalc(3, 7, 11, 15, 0x5, 0x1)
		Gprecalc(0, 5, 10, 15, 0xb, 0xf)
		Gprecalc(1, 6, 11, 12, 0xe, 0x9)
		Gprecalc(2, 7, 8, 13, 0xc, 0x3)
		Gprecalc(3, 4, 9, 14, 0x0, 0xd)

		Gprecalc(0, 4, 8, 12, 0x1, 0x0)
		Gprecalc(1, 5, 9, 13, 0x3, 0x2)
		Gprecalc(2, 6, 10, 14, 0x5, 0x4)
		Gprecalc(3, 7, 11, 15, 0x7, 0x6)
		Gprecalc(0, 5, 10, 15, 0x9, 0x8)
		Gprecalc(1, 6, 11, 12, 0xb, 0xa)
		Gprecalc(2, 7, 8, 13, 0xd, 0xc)
		Gprecalc(3, 4, 9, 14, 0xf, 0xe)

		Gprecalc(0, 4, 8, 12, 0xa, 0xe)
		Gprecalc(1, 5, 9, 13, 0x8, 0x4)
		Gprecalc(2, 6, 10, 14, 0xf, 0x9)
		Gprecalc(3, 7, 11, 15, 0x6, 0xd)
		Gprecalc(0, 5, 10, 15, 0xc, 0x1)
		Gprecalc(1, 6, 11, 12, 0x2, 0x0)
		Gprecalc(2, 7, 8, 13, 0x7, 0xb)
		Gprecalc(3, 4, 9, 14, 0x3, 0x5)

		Gprecalc(0, 4, 8, 12, 0x8, 0xb)
		Gprecalc(1, 5, 9, 13, 0x0, 0xc)
		Gprecalc(2, 6, 10, 14, 0x2, 0x5)
		Gprecalc(3, 7, 11, 15, 0xd, 0xf)
		Gprecalc(0, 5, 10, 15, 0xe, 0xa)
		Gprecalc(1, 6, 11, 12, 0x6, 0x3)
		Gprecalc(2, 7, 8, 13, 0x1, 0x7)
		Gprecalc(3, 4, 9, 14, 0x4, 0x9)

		Gprecalc(0, 4, 8, 12, 0x9, 0x7)
		Gprecalc(1, 5, 9, 13, 0x1, 0x3)
		Gprecalc(2, 6, 10, 14, 0xc, 0xd)
		Gprecalc(3, 7, 11, 15, 0xe, 0xb)
		Gprecalc(0, 5, 10, 15, 0x6, 0x2)
		Gprecalc(1, 6, 11, 12, 0xa, 0x5)
		Gprecalc(2, 7, 8, 13, 0x0, 0x4)
		Gprecalc(3, 4, 9, 14, 0x8, 0xf)

		Gprecalc(0, 4, 8, 12, 0x0, 0x9)
		Gprecalc(1, 5, 9, 13, 0x7, 0x5)
		Gprecalc(2, 6, 10, 14, 0x4, 0x2)
		Gprecalc(3, 7, 11, 15, 0xf, 0xa)
		Gprecalc(0, 5, 10, 15, 0x1, 0xe)
		Gprecalc(1, 6, 11, 12, 0xc, 0xb)
		Gprecalc(2, 7, 8, 13, 0x8, 0x6)
		Gprecalc(3, 4, 9, 14, 0xd, 0x3)

		Gprecalc(0, 4, 8, 12, 0xc, 0x2)
		Gprecalc(1, 5, 9, 13, 0xa, 0x6)
		Gprecalc(2, 6, 10, 14, 0xb, 0x0)
		Gprecalc(3, 7, 11, 15, 0x3, 0x8)
		Gprecalc(0, 5, 10, 15, 0xd, 0x4)
		Gprecalc(1, 6, 11, 12, 0x5, 0x7)
		Gprecalc(2, 7, 8, 13, 0xe, 0xf)
		Gprecalc(3, 4, 9, 14, 0x9, 0x1)

		uint64_t *outHash = &g_hash[8 * hashPosition];

		outHash[0] = devectorizeswap(h[0] ^ v[0] ^ v[8]);
		outHash[1] = devectorizeswap(h[1] ^ v[1] ^ v[9]);
		outHash[2] = devectorizeswap(h[2] ^ v[2] ^ v[10]);
		outHash[3] = devectorizeswap(h[3] ^ v[3] ^ v[11]);
		outHash[4] = devectorizeswap(h[4] ^ v[4] ^ v[12]);
		outHash[5] = devectorizeswap(h[5] ^ v[5] ^ v[13]);
		outHash[6] = devectorizeswap(h[6] ^ v[6] ^ v[14]);
		outHash[7] = devectorizeswap(h[7] ^ v[7] ^ v[15]);
	}
}

__global__ 
#if __CUDA_ARCH__ > 500
__launch_bounds__(256, 4)
#else
__launch_bounds__(32, 32)
#endif
void quark_blake512_gpu_hash_80(uint32_t threads, uint32_t startNounce, uint32_t *outputHash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t nounce = startNounce + thread;

		uint2 block[16];

		// Message f�r die erste Runde in Register holen
#pragma unroll 16
		for (int i = 0; i < 16; ++i)
			block[i] = vectorize(c_PaddedMessage80[i]);
		// The test Nonce
			//		((uint32_t*)block)[18] = nounce;
		block[9].x = nounce;
//		((uint32_t*)block)[18] = nounce;

		const uint2 u512[16] =
		{
			{ 0x85a308d3UL, 0x243f6a88 }, { 0x03707344UL, 0x13198a2e },
			{ 0x299f31d0UL, 0xa4093822 }, { 0xec4e6c89UL, 0x082efa98 },
			{ 0x38d01377UL, 0x452821e6 }, { 0x34e90c6cUL, 0xbe5466cf },
			{ 0xc97c50ddUL, 0xc0ac29b7 }, { 0xb5470917UL, 0x3f84d5b5 },
			{ 0x8979fb1bUL, 0x9216d5d9 }, { 0x98dfb5acUL, 0xd1310ba6 },
			{ 0xd01adfb7UL, 0x2ffd72db }, { 0x6a267e96UL, 0xb8e1afed },
			{ 0xf12c7f99UL, 0xba7c9045 }, { 0xb3916cf7UL, 0x24a19947 },
			{ 0x858efc16UL, 0x0801f2e2 }, { 0x71574e69UL, 0x636920d8 }
		};

		const uint2 h[8] = {
				{ 0xf3bcc908UL,0x6a09e667UL },
				{ 0x84caa73bUL ,0xbb67ae85UL },
				{ 0xfe94f82bUL,0x3c6ef372UL },
				{ 0x5f1d36f1UL,0xa54ff53aUL },
				{ 0xade682d1UL,0x510e527fUL },
				{ 0x2b3e6c1fUL,0x9b05688cUL },
				{ 0xfb41bd6bUL,0x1f83d9abUL },
				{ 0x137e2179UL,0x5be0cd19UL }
		};

		uint2 v[16] =
		{
			h[0], h[1], h[2], h[3], h[4], h[5], h[6], h[7],
			u512[0], u512[1], u512[2], u512[3], u512[4] ^ 640, u512[5] ^ 640, u512[6], u512[7]
		};

		Gprecalc(0, 4, 8, 12, 0x1, 0x0)
		Gprecalc(1, 5, 9, 13, 0x3, 0x2)
		Gprecalc(2, 6, 10, 14, 0x5, 0x4)
		Gprecalc(3, 7, 11, 15, 0x7, 0x6)
		Gprecalc(0, 5, 10, 15, 0x9, 0x8)
		Gprecalc(1, 6, 11, 12, 0xb, 0xa)
		Gprecalc(2, 7, 8, 13, 0xd, 0xc)
		Gprecalc(3, 4, 9, 14, 0xf, 0xe)

		Gprecalc(0, 4, 8, 12, 0xa, 0xe)
		Gprecalc(1, 5, 9, 13, 0x8, 0x4)
		Gprecalc(2, 6, 10, 14, 0xf, 0x9)
		Gprecalc(3, 7, 11, 15, 0x6, 0xd)
		Gprecalc(0, 5, 10, 15, 0xc, 0x1)
		Gprecalc(1, 6, 11, 12, 0x2, 0x0)
		Gprecalc(2, 7, 8, 13, 0x7, 0xb)
		Gprecalc(3, 4, 9, 14, 0x3, 0x5)

		Gprecalc(0, 4, 8, 12, 0x8, 0xb)
		Gprecalc(1, 5, 9, 13, 0x0, 0xc)
		Gprecalc(2, 6, 10, 14, 0x2, 0x5)
		Gprecalc(3, 7, 11, 15, 0xd, 0xf)
		Gprecalc(0, 5, 10, 15, 0xe, 0xa)
		Gprecalc(1, 6, 11, 12, 0x6, 0x3)
		Gprecalc(2, 7, 8, 13, 0x1, 0x7)
		Gprecalc(3, 4, 9, 14, 0x4, 0x9)

		Gprecalc(0, 4, 8, 12, 0x9, 0x7)
		Gprecalc(1, 5, 9, 13, 0x1, 0x3)
		Gprecalc(2, 6, 10, 14, 0xc, 0xd)
		Gprecalc(3, 7, 11, 15, 0xe, 0xb)
		Gprecalc(0, 5, 10, 15, 0x6, 0x2)
		Gprecalc(1, 6, 11, 12, 0xa, 0x5)
		Gprecalc(2, 7, 8, 13, 0x0, 0x4)
		Gprecalc(3, 4, 9, 14, 0x8, 0xf)

		Gprecalc(0, 4, 8, 12, 0x0, 0x9)
		Gprecalc(1, 5, 9, 13, 0x7, 0x5)
		Gprecalc(2, 6, 10, 14, 0x4, 0x2)
		Gprecalc(3, 7, 11, 15, 0xf, 0xa)
		Gprecalc(0, 5, 10, 15, 0x1, 0xe)
		Gprecalc(1, 6, 11, 12, 0xc, 0xb)
		Gprecalc(2, 7, 8, 13, 0x8, 0x6)
		Gprecalc(3, 4, 9, 14, 0xd, 0x3)
		
		Gprecalc(0, 4, 8, 12, 0xc, 0x2)
		Gprecalc(1, 5, 9, 13, 0xa, 0x6)
		Gprecalc(2, 6, 10, 14, 0xb, 0x0)
		Gprecalc(3, 7, 11, 15, 0x3, 0x8)
		Gprecalc(0, 5, 10, 15, 0xd, 0x4)
		Gprecalc(1, 6, 11, 12, 0x5, 0x7)
		Gprecalc(2, 7, 8, 13, 0xe, 0xf)
		Gprecalc(3, 4, 9, 14, 0x9, 0x1)

		Gprecalc(0, 4, 8, 12, 0x5, 0xc)
		Gprecalc(1, 5, 9, 13, 0xf, 0x1)
		Gprecalc(2, 6, 10, 14, 0xd, 0xe)
		Gprecalc(3, 7, 11, 15, 0xa, 0x4)
		Gprecalc(0, 5, 10, 15, 0x7, 0x0)
		Gprecalc(1, 6, 11, 12, 0x3, 0x6)
		Gprecalc(2, 7, 8, 13, 0x2, 0x9)
		Gprecalc(3, 4, 9, 14, 0xb, 0x8)

		Gprecalc(0, 4, 8, 12, 0xb, 0xd)
		Gprecalc(1, 5, 9, 13, 0xe, 0x7)
		Gprecalc(2, 6, 10, 14, 0x1, 0xc)
		Gprecalc(3, 7, 11, 15, 0x9, 0x3)
		Gprecalc(0, 5, 10, 15, 0x0, 0x5)
		Gprecalc(1, 6, 11, 12, 0x4, 0xf)
		Gprecalc(2, 7, 8, 13, 0x6, 0x8)
		Gprecalc(3, 4, 9, 14, 0xa, 0x2)

		Gprecalc(0, 4, 8, 12, 0xf, 0x6)
		Gprecalc(1, 5, 9, 13, 0x9, 0xe)
		Gprecalc(2, 6, 10, 14, 0x3, 0xb)
		Gprecalc(3, 7, 11, 15, 0x8, 0x0)
		Gprecalc(0, 5, 10, 15, 0x2, 0xc)
		Gprecalc(1, 6, 11, 12, 0x7, 0xd)
		Gprecalc(2, 7, 8, 13, 0x4, 0x1)
		Gprecalc(3, 4, 9, 14, 0x5, 0xa)

		Gprecalc(0, 4, 8, 12, 0x2, 0xa)
		Gprecalc(1, 5, 9, 13, 0x4, 0x8)
		Gprecalc(2, 6, 10, 14, 0x6, 0x7)
		Gprecalc(3, 7, 11, 15, 0x5, 0x1)
		Gprecalc(0, 5, 10, 15, 0xb, 0xf)
		Gprecalc(1, 6, 11, 12, 0xe, 0x9)
		Gprecalc(2, 7, 8, 13, 0xc, 0x3)
		Gprecalc(3, 4, 9, 14, 0x0, 0xd)
		
		Gprecalc(0, 4, 8, 12, 0x1, 0x0)
		Gprecalc(1, 5, 9, 13, 0x3, 0x2)
		Gprecalc(2, 6, 10, 14, 0x5, 0x4)
		Gprecalc(3, 7, 11, 15, 0x7, 0x6)
		Gprecalc(0, 5, 10, 15, 0x9, 0x8)
		Gprecalc(1, 6, 11, 12, 0xb, 0xa)
		Gprecalc(2, 7, 8, 13, 0xd, 0xc)
		Gprecalc(3, 4, 9, 14, 0xf, 0xe)

		Gprecalc(0, 4, 8, 12, 0xa, 0xe)
		Gprecalc(1, 5, 9, 13, 0x8, 0x4)
		Gprecalc(2, 6, 10, 14, 0xf, 0x9)
		Gprecalc(3, 7, 11, 15, 0x6, 0xd)
		Gprecalc(0, 5, 10, 15, 0xc, 0x1)
		Gprecalc(1, 6, 11, 12, 0x2, 0x0)
		Gprecalc(2, 7, 8, 13, 0x7, 0xb)
		Gprecalc(3, 4, 9, 14, 0x3, 0x5)

		Gprecalc(0, 4, 8, 12, 0x8, 0xb)
		Gprecalc(1, 5, 9, 13, 0x0, 0xc)
		Gprecalc(2, 6, 10, 14, 0x2, 0x5)
		Gprecalc(3, 7, 11, 15, 0xd, 0xf)
		Gprecalc(0, 5, 10, 15, 0xe, 0xa)
		Gprecalc(1, 6, 11, 12, 0x6, 0x3)
		Gprecalc(2, 7, 8, 13, 0x1, 0x7)
		Gprecalc(3, 4, 9, 14, 0x4, 0x9)

		Gprecalc(0, 4, 8, 12, 0x9, 0x7)
		Gprecalc(1, 5, 9, 13, 0x1, 0x3)
		Gprecalc(2, 6, 10, 14, 0xc, 0xd)
		Gprecalc(3, 7, 11, 15, 0xe, 0xb)
		Gprecalc(0, 5, 10, 15, 0x6, 0x2)
		Gprecalc(1, 6, 11, 12, 0xa, 0x5)
		Gprecalc(2, 7, 8, 13, 0x0, 0x4)
		Gprecalc(3, 4, 9, 14, 0x8, 0xf)

		Gprecalc(0, 4, 8, 12, 0x0, 0x9)
		Gprecalc(1, 5, 9, 13, 0x7, 0x5)
		Gprecalc(2, 6, 10, 14, 0x4, 0x2)
		Gprecalc(3, 7, 11, 15, 0xf, 0xa)
		Gprecalc(0, 5, 10, 15, 0x1, 0xe)
		Gprecalc(1, 6, 11, 12, 0xc, 0xb)
		Gprecalc(2, 7, 8, 13, 0x8, 0x6)
		Gprecalc(3, 4, 9, 14, 0xd, 0x3)
		
		Gprecalc(0, 4, 8, 12, 0xc, 0x2)
		Gprecalc(1, 5, 9, 13, 0xa, 0x6)
		Gprecalc(2, 6, 10, 14, 0xb, 0x0)
		Gprecalc(3, 7, 11, 15, 0x3, 0x8)
		Gprecalc(0, 5, 10, 15, 0xd, 0x4)
		Gprecalc(1, 6, 11, 12, 0x5, 0x7)
		Gprecalc(2, 7, 8, 13, 0xe, 0xf)
		Gprecalc(3, 4, 9, 14, 0x9, 0x1)

		uint64_t *outHash = (uint64_t *)outputHash + 8 * thread;
		outHash[0] = devectorizeswap(h[0] ^ v[0] ^ v[8]);
		outHash[1] = devectorizeswap(h[1] ^ v[1] ^ v[9]);
		outHash[2] = devectorizeswap(h[2] ^ v[2] ^ v[10]);
		outHash[3] = devectorizeswap(h[3] ^ v[3] ^ v[11]);
		outHash[4] = devectorizeswap(h[4] ^ v[4] ^ v[12]);
		outHash[5] = devectorizeswap(h[5] ^ v[5] ^ v[13]);
		outHash[6] = devectorizeswap(h[6] ^ v[6] ^ v[14]);
		outHash[7] = devectorizeswap(h[7] ^ v[7] ^ v[15]);
	}
}


// ---------------------------- END CUDA quark_blake512 functions ------------------------------------


// Blake512 f�r 80 Byte grosse Eingangsdaten
__host__ void quark_blake512_cpu_setBlock_80(void *pdata)
{
	// Message mit Padding bereitstellen
	// lediglich die korrekte Nonce ist noch ab Byte 76 einzusetzen.
	unsigned char PaddedMessage[128];
	memcpy(PaddedMessage, pdata, 80);
	memset(PaddedMessage+80, 0, 48);
	PaddedMessage[80] = 0x80;
	PaddedMessage[111] = 1;
	PaddedMessage[126] = 0x02;
	PaddedMessage[127] = 0x80;
	for (int i = 0; i < 16; i++)
		((uint64_t*)PaddedMessage)[i] = cuda_swab64(((uint64_t*)PaddedMessage)[i]);
	CUDA_SAFE_CALL(
		hipMemcpyToSymbol(HIP_SYMBOL(c_PaddedMessage80), PaddedMessage, 16*sizeof(uint64_t), 0, hipMemcpyHostToDevice)
	);
}


__host__ void quark_blake512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_outputHash, int order)
{
	const uint32_t threadsperblock = 32;
	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);
	quark_blake512_gpu_hash_64<<<grid, block>>>(threads, startNounce, d_nonceVector, (uint64_t*)d_outputHash);
}

__host__ void quark_blake512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_outputHash, int order)
{

	const uint32_t threadsperblock = 32;
	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	quark_blake512_gpu_hash_80<<<grid, block>>>(threads, startNounce, d_outputHash);
//	MyStreamSynchronize(NULL, order, thr_id);
}
