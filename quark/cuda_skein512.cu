#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <memory.h>

#include "cuda_helper.h"
#define TPB 128 
#define TPBf 128

// Take a look at: https://www.schneier.com/skein1.3.pdf

#define SHL(x, n)			((x) << (n))
#define SHR(x, n)			((x) >> (n))

static uint32_t *d_nonce[MAX_GPUS];

/*
 * M9_ ## s ## _ ## i  evaluates to s+i mod 9 (0 <= s <= 18, 0 <= i <= 7).
 */

#define M9_0_0    0
#define M9_0_1    1
#define M9_0_2    2
#define M9_0_3    3
#define M9_0_4    4
#define M9_0_5    5
#define M9_0_6    6
#define M9_0_7    7

#define M9_1_0    1
#define M9_1_1    2
#define M9_1_2    3
#define M9_1_3    4
#define M9_1_4    5
#define M9_1_5    6
#define M9_1_6    7
#define M9_1_7    8

#define M9_2_0    2
#define M9_2_1    3
#define M9_2_2    4
#define M9_2_3    5
#define M9_2_4    6
#define M9_2_5    7
#define M9_2_6    8
#define M9_2_7    0

#define M9_3_0    3
#define M9_3_1    4
#define M9_3_2    5
#define M9_3_3    6
#define M9_3_4    7
#define M9_3_5    8
#define M9_3_6    0
#define M9_3_7    1

#define M9_4_0    4
#define M9_4_1    5
#define M9_4_2    6
#define M9_4_3    7
#define M9_4_4    8
#define M9_4_5    0
#define M9_4_6    1
#define M9_4_7    2

#define M9_5_0    5
#define M9_5_1    6
#define M9_5_2    7
#define M9_5_3    8
#define M9_5_4    0
#define M9_5_5    1
#define M9_5_6    2
#define M9_5_7    3

#define M9_6_0    6
#define M9_6_1    7
#define M9_6_2    8
#define M9_6_3    0
#define M9_6_4    1
#define M9_6_5    2
#define M9_6_6    3
#define M9_6_7    4

#define M9_7_0    7
#define M9_7_1    8
#define M9_7_2    0
#define M9_7_3    1
#define M9_7_4    2
#define M9_7_5    3
#define M9_7_6    4
#define M9_7_7    5

#define M9_8_0    8
#define M9_8_1    0
#define M9_8_2    1
#define M9_8_3    2
#define M9_8_4    3
#define M9_8_5    4
#define M9_8_6    5
#define M9_8_7    6

#define M9_9_0    0
#define M9_9_1    1
#define M9_9_2    2
#define M9_9_3    3
#define M9_9_4    4
#define M9_9_5    5
#define M9_9_6    6
#define M9_9_7    7

#define M9_10_0   1
#define M9_10_1   2
#define M9_10_2   3
#define M9_10_3   4
#define M9_10_4   5
#define M9_10_5   6
#define M9_10_6   7
#define M9_10_7   8

#define M9_11_0   2
#define M9_11_1   3
#define M9_11_2   4
#define M9_11_3   5
#define M9_11_4   6
#define M9_11_5   7
#define M9_11_6   8
#define M9_11_7   0

#define M9_12_0   3
#define M9_12_1   4
#define M9_12_2   5
#define M9_12_3   6
#define M9_12_4   7
#define M9_12_5   8
#define M9_12_6   0
#define M9_12_7   1

#define M9_13_0   4
#define M9_13_1   5
#define M9_13_2   6
#define M9_13_3   7
#define M9_13_4   8
#define M9_13_5   0
#define M9_13_6   1
#define M9_13_7   2

#define M9_14_0   5
#define M9_14_1   6
#define M9_14_2   7
#define M9_14_3   8
#define M9_14_4   0
#define M9_14_5   1
#define M9_14_6   2
#define M9_14_7   3

#define M9_15_0   6
#define M9_15_1   7
#define M9_15_2   8
#define M9_15_3   0
#define M9_15_4   1
#define M9_15_5   2
#define M9_15_6   3
#define M9_15_7   4

#define M9_16_0   7
#define M9_16_1   8
#define M9_16_2   0
#define M9_16_3   1
#define M9_16_4   2
#define M9_16_5   3
#define M9_16_6   4
#define M9_16_7   5

#define M9_17_0   8
#define M9_17_1   0
#define M9_17_2   1
#define M9_17_3   2
#define M9_17_4   3
#define M9_17_5   4
#define M9_17_6   5
#define M9_17_7   6

#define M9_18_0   0
#define M9_18_1   1
#define M9_18_2   2
#define M9_18_3   3
#define M9_18_4   4
#define M9_18_5   5
#define M9_18_6   6
#define M9_18_7   7

/*
 * M3_ ## s ## _ ## i  evaluates to s+i mod 3 (0 <= s <= 18, 0 <= i <= 1).
 */

#define M3_0_0    0
#define M3_0_1    1
#define M3_1_0    1
#define M3_1_1    2
#define M3_2_0    2
#define M3_2_1    0
#define M3_3_0    0
#define M3_3_1    1
#define M3_4_0    1
#define M3_4_1    2
#define M3_5_0    2
#define M3_5_1    0
#define M3_6_0    0
#define M3_6_1    1
#define M3_7_0    1
#define M3_7_1    2
#define M3_8_0    2
#define M3_8_1    0
#define M3_9_0    0
#define M3_9_1    1
#define M3_10_0   1
#define M3_10_1   2
#define M3_11_0   2
#define M3_11_1   0
#define M3_12_0   0
#define M3_12_1   1
#define M3_13_0   1
#define M3_13_1   2
#define M3_14_0   2
#define M3_14_1   0
#define M3_15_0   0
#define M3_15_1   1
#define M3_16_0   1
#define M3_16_1   2
#define M3_17_0   2
#define M3_17_1   0
#define M3_18_0   0
#define M3_18_1   1

#define XCAT(x, y)     XCAT_(x, y)
#define XCAT_(x, y)    x ## y

#define SKBI(k, s, i)   XCAT(k, XCAT(XCAT(XCAT(M9_, s), _), i))
#define SKBT(t, s, v)   XCAT(t, XCAT(XCAT(XCAT(M3_, s), _), v))

#define TFBIG_KINIT(k0, k1, k2, k3, k4, k5, k6, k7, k8, t0, t1, t2) { \
		k8 = ((k0 ^ k1) ^ (k2 ^ k3)) ^ ((k4 ^ k5) ^ (k6 ^ k7)) \
			^ make_uint2( 0xA9FC1A22UL,0x1BD11BDA); \
		t2 = t0 ^ t1; \
	}
//vectorize(0x1BD11BDAA9FC1A22ULL);
#define TFBIG_ADDKEY(w0, w1, w2, w3, w4, w5, w6, w7, k, t, s) { \
		w0 = (w0 + SKBI(k, s, 0)); \
		w1 = (w1 + SKBI(k, s, 1)); \
		w2 = (w2 + SKBI(k, s, 2)); \
		w3 = (w3 + SKBI(k, s, 3)); \
		w4 = (w4 + SKBI(k, s, 4)); \
		w5 = (w5 + SKBI(k, s, 5) + SKBT(t, s, 0)); \
		w6 = (w6 + SKBI(k, s, 6) + SKBT(t, s, 1)); \
		w7 = (w7 + SKBI(k, s, 7) + vectorizelow(s)); \
	}

#define TFBIG_MIX(x0, x1, rc) { \
		x0 = x0 + x1; \
		x1 = ROL2(x1, rc) ^ x0; \
	}

#define TFBIG_MIX8(w0, w1, w2, w3, w4, w5, w6, w7, rc0, rc1, rc2, rc3) { \
		TFBIG_MIX(w0, w1, rc0); \
		TFBIG_MIX(w2, w3, rc1); \
		TFBIG_MIX(w4, w5, rc2); \
		TFBIG_MIX(w6, w7, rc3); \
	}

#define TFBIG_4e(s)  { \
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, s); \
		TFBIG_MIX8(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], 46, 36, 19, 37); \
		TFBIG_MIX8(p[2], p[1], p[4], p[7], p[6], p[5], p[0], p[3], 33, 27, 14, 42); \
		TFBIG_MIX8(p[4], p[1], p[6], p[3], p[0], p[5], p[2], p[7], 17, 49, 36, 39); \
		TFBIG_MIX8(p[6], p[1], p[0], p[7], p[2], p[5], p[4], p[3], 44,  9, 54, 56); \
	}

#define TFBIG_4o(s)  { \
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, s); \
		TFBIG_MIX8(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], 39, 30, 34, 24); \
		TFBIG_MIX8(p[2], p[1], p[4], p[7], p[6], p[5], p[0], p[3], 13, 50, 10, 17); \
		TFBIG_MIX8(p[4], p[1], p[6], p[3], p[0], p[5], p[2], p[7], 25, 29, 39, 43); \
		TFBIG_MIX8(p[6], p[1], p[0], p[7], p[2], p[5], p[4], p[3],  8, 35, 56, 22); \
	}

__global__
#if __CUDA_ARCH__ > 500
__launch_bounds__(TPB, 2)
#else
__launch_bounds__(TPB, 1)
#endif
void quark_skein512_gpu_hash_64(uint32_t threads, uint32_t startNounce, uint64_t * const __restrict__ g_hash, uint32_t *g_nonceVector)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		// Skein
		uint2 p[8];
		uint2 h0, h1, h2, h3, h4, h5, h6, h7, h8;
		uint2 t0, t1, t2;

		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

		int hashPosition = nounce - startNounce;
		uint64_t *inpHash = &g_hash[8 * hashPosition];

		h0 = make_uint2(0x749C51CEull, 0x4903ADFF);
		h1 = make_uint2(0x9746DF03ull, 0x0D95DE39);
		h2 = make_uint2(0x27C79BCEull, 0x8FD19341);
		h3 = make_uint2(0xFF352CB1ull, 0x9A255629);
		h4 = make_uint2(0xDF6CA7B0ull, 0x5DB62599);
		h5 = make_uint2(0xA9D5C3F4ull, 0xEABE394C);
		h6 = make_uint2(0x1A75B523ull, 0x991112C7);
		h7 = make_uint2(0x660FCC33ull, 0xAE18A40B);

		// 1. Runde -> etype = 480, ptr = 64, bcount = 0, data = msg		
#pragma unroll 8
		for(int i=0;i<8;i++)
			p[i] = vectorize(inpHash[i]);

		t0 = vectorizelow(64); // ptr
		t1 = vectorize(480ull << 55); // etype
		TFBIG_KINIT(h0, h1, h2, h3, h4, h5, h6, h7, h8, t0, t1, t2);
		TFBIG_4e(0);
		TFBIG_4o(1);
		TFBIG_4e(2);
		TFBIG_4o(3);
		TFBIG_4e(4);
		TFBIG_4o(5);
		TFBIG_4e(6);
		TFBIG_4o(7);
		TFBIG_4e(8);
		TFBIG_4o(9);
		TFBIG_4e(10);
		TFBIG_4o(11);
		TFBIG_4e(12);
		TFBIG_4o(13);
		TFBIG_4e(14);
		TFBIG_4o(15);
		TFBIG_4e(16);
		TFBIG_4o(17);
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, 18);

		h0 = vectorize(inpHash[0]) ^ p[0];
		h1 = vectorize(inpHash[1]) ^ p[1];
		h2 = vectorize(inpHash[2]) ^ p[2];
		h3 = vectorize(inpHash[3]) ^ p[3];
		h4 = vectorize(inpHash[4]) ^ p[4];
		h5 = vectorize(inpHash[5]) ^ p[5];
		h6 = vectorize(inpHash[6]) ^ p[6];
		h7 = vectorize(inpHash[7]) ^ p[7];

		// 2. Runde -> etype = 510, ptr = 8, bcount = 0, data = 0
#pragma unroll 8
		for(int i=0;i<8;i++)
			p[i] = make_uint2(0,0);

		t0 = vectorizelow(8); // ptr
		t1 = vectorize(510ull << 55); // etype
		TFBIG_KINIT(h0, h1, h2, h3, h4, h5, h6, h7, h8, t0, t1, t2);
		TFBIG_4e(0);
		TFBIG_4o(1);
		TFBIG_4e(2);
		TFBIG_4o(3);
		TFBIG_4e(4);
		TFBIG_4o(5);
		TFBIG_4e(6);
		TFBIG_4o(7);
		TFBIG_4e(8);
		TFBIG_4o(9);
		TFBIG_4e(10);
		TFBIG_4o(11);
		TFBIG_4e(12);
		TFBIG_4o(13);
		TFBIG_4e(14);
		TFBIG_4o(15);
		TFBIG_4e(16);
		TFBIG_4o(17);
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, 18);

		// fertig
		uint64_t *outpHash = &g_hash[8 * hashPosition];

#pragma unroll 8
		for(int i=0;i<8;i++)
			outpHash[i] = devectorize(p[i]);
	}
}

__global__ 
#if __CUDA_ARCH__ > 500
__launch_bounds__(TPBf, 2)
#else
__launch_bounds__(TPBf, 1)
#endif
void quark_skein512_gpu_hash_64_final(const uint32_t threads, const uint32_t startNounce, uint64_t * const __restrict__ g_hash, const uint32_t *g_nonceVector, uint32_t *d_nonce, uint32_t target)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		// Skein
		uint2 p[8];
		uint2 h0, h1, h2, h3, h4, h5, h6, h7, h8;
		uint2 t0, t1, t2;

		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

		int hashPosition = nounce - startNounce;
		uint64_t *inpHash = &g_hash[8 * hashPosition];

		h0 = make_uint2(0x749C51CEull, 0x4903ADFF);
		h1 = make_uint2(0x9746DF03ull, 0x0D95DE39);
		h2 = make_uint2(0x27C79BCEull, 0x8FD19341);
		h3 = make_uint2(0xFF352CB1ull, 0x9A255629);
		h4 = make_uint2(0xDF6CA7B0ull, 0x5DB62599);
		h5 = make_uint2(0xA9D5C3F4ull, 0xEABE394C);
		h6 = make_uint2(0x1A75B523ull, 0x991112C7);
		h7 = make_uint2(0x660FCC33ull, 0xAE18A40B);

		// 1. Runde -> etype = 480, ptr = 64, bcount = 0, data = msg		
#pragma unroll 8
		for (int i = 0; i<8; i++)
			p[i] = vectorize(inpHash[i]);

		t0 = vectorizelow(64); // ptr
		t1 = vectorize(480ull << 55); // etype
		TFBIG_KINIT(h0, h1, h2, h3, h4, h5, h6, h7, h8, t0, t1, t2);
		TFBIG_4e(0);
		TFBIG_4o(1);
		TFBIG_4e(2);
		TFBIG_4o(3);
		TFBIG_4e(4);
		TFBIG_4o(5);
		TFBIG_4e(6);
		TFBIG_4o(7);
		TFBIG_4e(8);
		TFBIG_4o(9);
		TFBIG_4e(10);
		TFBIG_4o(11);
		TFBIG_4e(12);
		TFBIG_4o(13);
		TFBIG_4e(14);
		TFBIG_4o(15);
		TFBIG_4e(16);
		TFBIG_4o(17);
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, 18);

		h0 = vectorize(inpHash[0]) ^ p[0];
		h1 = vectorize(inpHash[1]) ^ p[1];
		h2 = vectorize(inpHash[2]) ^ p[2];
		h3 = vectorize(inpHash[3]) ^ p[3];
		h4 = vectorize(inpHash[4]) ^ p[4];
		h5 = vectorize(inpHash[5]) ^ p[5];
		h6 = vectorize(inpHash[6]) ^ p[6];
		h7 = vectorize(inpHash[7]) ^ p[7];

		// 2. Runde -> etype = 510, ptr = 8, bcount = 0, data = 0
#pragma unroll 8
		for (int i = 0; i<8; i++)
			p[i] = make_uint2(0, 0);

		t0 = vectorizelow(8); // ptr
		t1 = vectorize(510ull << 55); // etype
		TFBIG_KINIT(h0, h1, h2, h3, h4, h5, h6, h7, h8, t0, t1, t2);
		TFBIG_4e(0);
		TFBIG_4o(1);
		TFBIG_4e(2);
		TFBIG_4o(3);
		TFBIG_4e(4);
		TFBIG_4o(5);
		TFBIG_4e(6);
		TFBIG_4o(7);
		TFBIG_4e(8);
		TFBIG_4o(9);
		TFBIG_4e(10);
		TFBIG_4o(11);
		TFBIG_4e(12);
		TFBIG_4o(13);
		TFBIG_4e(14);
		TFBIG_4o(15);
		TFBIG_4e(16);
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, 17); 
		p[0] = p[0] + p[1];
		p[1] = ROL2(p[1], 39) ^ p[0];
		p[2] = p[2] + p[3];
		p[3] = ROL2(p[3], 30) ^ p[2];
		p[4] = p[4] + p[5];
		p[5] = ROL2(p[5], 34) ^ p[4];
		p[6] = p[6] + p[7];
		p[7] = ROL2(p[7], 24) ^ p[6];
		p[1] = ROL2(p[1], 13) ^ (p[2] + p[1]);
		p[3] = ROL2(p[3], 17) ^ (p[0] + p[3]);
		p[3] = ROL2(p[3], 29) ^ (p[6] + p[5] + p[3]);
		p[3] = (ROL2(p[3], 22) ^ (p[4] + p[7] + p[1] + p[3])) + h3;

		if (p[3].y <= target)
		{
			uint32_t tmp = atomicExch(&d_nonce[0], nounce);
			if (tmp != 0xffffffff)
				d_nonce[1] = tmp;
		}
	}
}


__host__ void quark_skein512_cpu_init(int thr_id)
{
	hipMalloc(&d_nonce[thr_id], 2*sizeof(uint32_t));
}

__host__ void quark_skein512_setTarget(const void *ptarget)
{
}
__host__ void quark_skein512_cpu_free(int32_t thr_id)
{
	hipHostFree(&d_nonce[thr_id]);
}

__host__
void quark_skein512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + TPB-1)/TPB);
	dim3 block(TPB);

	quark_skein512_gpu_hash_64 << <grid, block>> >(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);
//	MyStreamSynchronize(NULL, order, thr_id);
}


__host__
void quark_skein512_cpu_hash_64_final(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, uint32_t *h_nonce, uint32_t target, int order)
{
	dim3 grid((threads + TPBf - 1) / TPBf);
	dim3 block(TPBf);

	hipMemset(d_nonce[thr_id], 0xff, 2*sizeof(uint32_t));

	quark_skein512_gpu_hash_64_final<< <grid, block>> >(threads, startNounce, (uint64_t*)d_hash, d_nonceVector, d_nonce[thr_id], target);
	hipMemcpy(h_nonce, d_nonce[thr_id], 2*sizeof(uint32_t), hipMemcpyDeviceToHost);
}

