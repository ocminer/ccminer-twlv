#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"

#ifdef _MSC_VER
#define UINT2(x,y) { x, y }
#else
#define UINT2(x,y) (uint2) { x, y }
#endif

__constant__ uint2 c_keccak_round_constants35[24] = {
		{ 0x00000001ul, 0x00000000 }, { 0x00008082ul, 0x00000000 },
		{ 0x0000808aul, 0x80000000 }, { 0x80008000ul, 0x80000000 },
		{ 0x0000808bul, 0x00000000 }, { 0x80000001ul, 0x00000000 },
		{ 0x80008081ul, 0x80000000 }, { 0x00008009ul, 0x80000000 },
		{ 0x0000008aul, 0x00000000 }, { 0x00000088ul, 0x00000000 },
		{ 0x80008009ul, 0x00000000 }, { 0x8000000aul, 0x00000000 },
		{ 0x8000808bul, 0x00000000 }, { 0x0000008bul, 0x80000000 },
		{ 0x00008089ul, 0x80000000 }, { 0x00008003ul, 0x80000000 },
		{ 0x00008002ul, 0x80000000 }, { 0x00000080ul, 0x80000000 },
		{ 0x0000800aul, 0x00000000 }, { 0x8000000aul, 0x80000000 },
		{ 0x80008081ul, 0x80000000 }, { 0x00008080ul, 0x80000000 },
		{ 0x80000001ul, 0x00000000 }, { 0x80008008ul, 0x80000000 }
};

static __device__ __forceinline__ void
keccak_block_35(uint2 *s) {
	int i = 0;
	uint2 t[5], u[5], v, w;

	t[0] = s[0] ^ s[5];
	t[1] = s[1] ^ s[6];
	t[2] = s[2] ^ s[7];
	t[3] = s[3] ^ s[8];

	/* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
	u[0] = s[4] ^ ROL2(t[1], 1);
	u[1] = t[0] ^ ROL2(t[2], 1);
	u[2] = t[1] ^ ROL2(t[3], 1);
	u[3] = t[2] ^ ROL2(s[4], 1);
	u[4] = t[3] ^ ROL2(t[0], 1);

	/* theta: a[0,i], a[1,i], .. a[4,i] ^= d[i] */
	s[0] ^= u[0]; s[5] ^= u[0]; s[10] ^= u[0]; s[15] ^= u[0]; s[20] ^= u[0];
	s[1] ^= u[1]; s[6] ^= u[1]; s[11] ^= u[1]; s[16] ^= u[1]; s[21] ^= u[1];
	s[2] ^= u[2]; s[7] ^= u[2]; s[12] ^= u[2]; s[17] ^= u[2]; s[22] ^= u[2];
	s[3] ^= u[3]; s[8] ^= u[3]; s[13] ^= u[3]; s[18] ^= u[3]; s[23] ^= u[3];
	s[4] ^= u[4]; s[9] ^= u[4]; s[14] ^= u[4]; s[19] ^= u[4]; s[24] ^= u[4];

	/* rho pi: b[..] = rotl(a[..], ..) */
	v = s[1];
	s[1] = ROL2(s[6], 44);
	s[6] = ROL2(s[9], 20);
	s[9] = ROL2(s[22], 61);
	s[22] = ROL2(s[14], 39);
	s[14] = ROL2(s[20], 18);
	s[20] = ROL2(s[2], 62);
	s[2] = ROL2(s[12], 43);
	s[12] = ROL2(s[13], 25);
	s[13] = ROL2(s[19], 8);
	s[19] = ROL2(s[23], 56);
	s[23] = ROL2(s[15], 41);
	s[15] = ROL2(s[4], 27);
	s[4] = ROL2(s[24], 14);
	s[24] = ROL2(s[21], 2);
	s[21] = ROL2(s[8], 55);
	s[8] = ROL2(s[16], 45);
	s[16] = ROL2(s[5], 36);
	s[5] = ROL2(s[3], 28);
	s[3] = ROL2(s[18], 21);
	s[18] = ROL2(s[17], 15);
	s[17] = ROL2(s[11], 10);
	s[11] = ROL2(s[7], 6);
	s[7] = ROL2(s[10], 3);
	s[10] = ROL2(v, 1);

	/* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
	v = s[0]; w = s[1]; s[0] ^= (~w) & s[2]; s[1] ^= (~s[2]) & s[3]; s[2] ^= (~s[3]) & s[4]; s[3] ^= (~s[4]) & v; s[4] ^= (~v) & w;
	v = s[5]; w = s[6]; s[5] ^= (~w) & s[7]; s[6] ^= (~s[7]) & s[8]; s[7] ^= (~s[8]) & s[9]; s[8] ^= (~s[9]) & v; s[9] ^= (~v) & w;
	v = s[10]; w = s[11]; s[10] ^= (~w) & s[12]; s[11] ^= (~s[12]) & s[13]; s[12] ^= (~s[13]) & s[14]; s[13] ^= (~s[14]) & v; s[14] ^= (~v) & w;
	v = s[15]; w = s[16]; s[15] ^= (~w) & s[17]; s[16] ^= (~s[17]) & s[18]; s[17] ^= (~s[18]) & s[19]; s[18] ^= (~s[19]) & v; s[19] ^= (~v) & w;
	v = s[20]; w = s[21]; s[20] ^= (~w) & s[22]; s[21] ^= (~s[22]) & s[23]; s[22] ^= (~s[23]) & s[24]; s[23] ^= (~s[24]) & v; s[24] ^= (~v) & w;

	/* iota: a[0,0] ^= round constant */
	s[0] = s[0] ^ 1; //c_keccak_round_constants[0]);

	for (i = 1; i < 24; i++) {
		/* theta: c = a[0,i] ^ a[1,i] ^ .. a[4,i] */
		t[0] = s[0] ^ s[5] ^ s[10] ^ s[15] ^ s[20];
		t[1] = s[1] ^ s[6] ^ s[11] ^ s[16] ^ s[21];
		t[2] = s[2] ^ s[7] ^ s[12] ^ s[17] ^ s[22];
		t[3] = s[3] ^ s[8] ^ s[13] ^ s[18] ^ s[23];
		t[4] = s[4] ^ s[9] ^ s[14] ^ s[19] ^ s[24];

		/* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
		u[0] = t[4] ^ ROL2(t[1], 1);
		u[1] = t[0] ^ ROL2(t[2], 1);
		u[2] = t[1] ^ ROL2(t[3], 1);
		u[3] = t[2] ^ ROL2(t[4], 1);
		u[4] = t[3] ^ ROL2(t[0], 1);

		/* theta: a[0,i], a[1,i], .. a[4,i] ^= d[i] */
		s[0] ^= u[0]; s[5] ^= u[0]; s[10] ^= u[0]; s[15] ^= u[0]; s[20] ^= u[0];
		s[1] ^= u[1]; s[6] ^= u[1]; s[11] ^= u[1]; s[16] ^= u[1]; s[21] ^= u[1];
		s[2] ^= u[2]; s[7] ^= u[2]; s[12] ^= u[2]; s[17] ^= u[2]; s[22] ^= u[2];
		s[3] ^= u[3]; s[8] ^= u[3]; s[13] ^= u[3]; s[18] ^= u[3]; s[23] ^= u[3];
		s[4] ^= u[4]; s[9] ^= u[4]; s[14] ^= u[4]; s[19] ^= u[4]; s[24] ^= u[4];

		/* rho pi: b[..] = rotl(a[..], ..) */
		v = s[1];
		s[1] = ROL2(s[6], 44);
		s[6] = ROL2(s[9], 20);
		s[9] = ROL2(s[22], 61);
		s[22] = ROL2(s[14], 39);
		s[14] = ROL2(s[20], 18);
		s[20] = ROL2(s[2], 62);
		s[2] = ROL2(s[12], 43);
		s[12] = ROL2(s[13], 25);
		s[13] = ROL2(s[19], 8);
		s[19] = ROL2(s[23], 56);
		s[23] = ROL2(s[15], 41);
		s[15] = ROL2(s[4], 27);
		s[4] = ROL2(s[24], 14);
		s[24] = ROL2(s[21], 2);
		s[21] = ROL2(s[8], 55);
		s[8] = ROL2(s[16], 45);
		s[16] = ROL2(s[5], 36);
		s[5] = ROL2(s[3], 28);
		s[3] = ROL2(s[18], 21);
		s[18] = ROL2(s[17], 15);
		s[17] = ROL2(s[11], 10);
		s[11] = ROL2(s[7], 6);
		s[7] = ROL2(s[10], 3);
		s[10] = ROL2(v, 1);

		/* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
		v = s[0]; w = s[1]; s[0] ^= (~w) & s[2]; s[1] ^= (~s[2]) & s[3]; s[2] ^= (~s[3]) & s[4]; s[3] ^= (~s[4]) & v; s[4] ^= (~v) & w;
		v = s[5]; w = s[6]; s[5] ^= (~w) & s[7]; s[6] ^= (~s[7]) & s[8]; s[7] ^= (~s[8]) & s[9]; s[8] ^= (~s[9]) & v; s[9] ^= (~v) & w;
		v = s[10]; w = s[11]; s[10] ^= (~w) & s[12]; s[11] ^= (~s[12]) & s[13]; s[12] ^= (~s[13]) & s[14]; s[13] ^= (~s[14]) & v; s[14] ^= (~v) & w;
		v = s[15]; w = s[16]; s[15] ^= (~w) & s[17]; s[16] ^= (~s[17]) & s[18]; s[17] ^= (~s[18]) & s[19]; s[18] ^= (~s[19]) & v; s[19] ^= (~v) & w;
		v = s[20]; w = s[21]; s[20] ^= (~w) & s[22]; s[21] ^= (~s[22]) & s[23]; s[22] ^= (~s[23]) & s[24]; s[23] ^= (~s[24]) & v; s[24] ^= (~v) & w;

		/* iota: a[0,0] ^= round constant */
		s[0] ^= c_keccak_round_constants35[i];
	}
}

static __device__ __forceinline__ void
keccak_block_35_final(uint2 *s)
{
	int i = 0;
	uint2 t[5], u[5], v, w;

	t[0] = s[0] ^ s[5];
	t[1] = s[1] ^ s[6];
	t[2] = s[2] ^ s[7];
	t[3] = s[3] ^ s[8];

	/* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
	u[0] = s[4] ^ ROL2(t[1], 1);
	u[1] = t[0] ^ ROL2(t[2], 1);
	u[2] = t[1] ^ ROL2(t[3], 1);
	u[3] = t[2] ^ ROL2(s[4], 1);
	u[4] = t[3] ^ ROL2(t[0], 1);

	/* theta: a[0,i], a[1,i], .. a[4,i] ^= d[i] */
	s[0] ^= u[0]; s[5] ^= u[0]; s[10] = u[0]; s[15] = u[0]; s[20] = u[0];
	s[1] ^= u[1]; s[6] ^= u[1]; s[11] = u[1]; s[16] = u[1]; s[21] = u[1];
	s[2] ^= u[2]; s[7] ^= u[2]; s[12] = u[2]; s[17] = u[2]; s[22] = u[2];
	s[3] ^= u[3]; s[8] ^= u[3]; s[13] = u[3]; s[18] = u[3]; s[23] = u[3];
	s[4] ^= u[4]; s[9] = u[4]; s[14] = u[4]; s[19] = u[4]; s[24] = u[4];

	/* rho pi: b[..] = rotl(a[..], ..) */
	v = s[1];
	s[1] = ROL2(s[6], 44);
	s[6] = ROL2(s[9], 20);
	s[9] = ROL2(s[22], 61);
	s[22] = ROL2(s[14], 39);
	s[14] = ROL2(s[20], 18);
	s[20] = ROL2(s[2], 62);
	s[2] = ROL2(s[12], 43);
	s[12] = ROL2(s[13], 25);
	s[13] = ROL2(s[19], 8);
	s[19] = ROL2(s[23], 56);
	s[23] = ROL2(s[15], 41);
	s[15] = ROL2(s[4], 27);
	s[4] = ROL2(s[24], 14);
	s[24] = ROL2(s[21], 2);
	s[21] = ROL2(s[8], 55);
	s[8] = ROL2(s[16], 45);
	s[16] = ROL2(s[5], 36);
	s[5] = ROL2(s[3], 28);
	s[3] = ROL2(s[18], 21);
	s[18] = ROL2(s[17], 15);
	s[17] = ROL2(s[11], 10);
	s[11] = ROL2(s[7], 6);
	s[7] = ROL2(s[10], 3);
	s[10] = ROL2(v, 1);

	/* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
	v = s[0]; w = s[1]; s[0] ^= (~w) & s[2]; s[1] ^= (~s[2]) & s[3]; s[2] ^= (~s[3]) & s[4]; s[3] ^= (~s[4]) & v; s[4] ^= (~v) & w;
	v = s[5]; w = s[6]; s[5] ^= (~w) & s[7]; s[6] ^= (~s[7]) & s[8]; s[7] ^= (~s[8]) & s[9]; s[8] ^= (~s[9]) & v; s[9] ^= (~v) & w;
	v = s[10]; w = s[11]; s[10] ^= (~w) & s[12]; s[11] ^= (~s[12]) & s[13]; s[12] ^= (~s[13]) & s[14]; s[13] ^= (~s[14]) & v; s[14] ^= (~v) & w;
	v = s[15]; w = s[16]; s[15] ^= (~w) & s[17]; s[16] ^= (~s[17]) & s[18]; s[17] ^= (~s[18]) & s[19]; s[18] ^= (~s[19]) & v; s[19] ^= (~v) & w;
	v = s[20]; w = s[21]; s[20] ^= (~w) & s[22]; s[21] ^= (~s[22]) & s[23]; s[22] ^= (~s[23]) & s[24]; s[23] ^= (~s[24]) & v; s[24] ^= (~v) & w;

	/* iota: a[0,0] ^= round constant */
	s[0] = s[0] ^ 1; //c_keccak_round_constants[0]);

	for (i = 1; i < 23; i++)
	{
		/* theta: c = a[0,i] ^ a[1,i] ^ .. a[4,i] */
		t[0] = s[0] ^ s[5] ^ s[10] ^ s[15] ^ s[20];
		t[1] = s[1] ^ s[6] ^ s[11] ^ s[16] ^ s[21];
		t[2] = s[2] ^ s[7] ^ s[12] ^ s[17] ^ s[22];
		t[3] = s[3] ^ s[8] ^ s[13] ^ s[18] ^ s[23];
		t[4] = s[4] ^ s[9] ^ s[14] ^ s[19] ^ s[24];

		/* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
		u[0] = t[4] ^ ROL2(t[1], 1);
		u[1] = t[0] ^ ROL2(t[2], 1);
		u[2] = t[1] ^ ROL2(t[3], 1);
		u[3] = t[2] ^ ROL2(t[4], 1);
		u[4] = t[3] ^ ROL2(t[0], 1);

		/* theta: a[0,i], a[1,i], .. a[4,i] ^= d[i] */
		s[0] ^= u[0]; s[5] ^= u[0]; s[10] ^= u[0]; s[15] ^= u[0]; s[20] ^= u[0];
		s[1] ^= u[1]; s[6] ^= u[1]; s[11] ^= u[1]; s[16] ^= u[1]; s[21] ^= u[1];
		s[2] ^= u[2]; s[7] ^= u[2]; s[12] ^= u[2]; s[17] ^= u[2]; s[22] ^= u[2];
		s[3] ^= u[3]; s[8] ^= u[3]; s[13] ^= u[3]; s[18] ^= u[3]; s[23] ^= u[3];
		s[4] ^= u[4]; s[9] ^= u[4]; s[14] ^= u[4]; s[19] ^= u[4]; s[24] ^= u[4];

		/* rho pi: b[..] = rotl(a[..], ..) */
		v = s[1];
		s[1] = ROL2(s[6], 44);
		s[6] = ROL2(s[9], 20);
		s[9] = ROL2(s[22], 61);
		s[22] = ROL2(s[14], 39);
		s[14] = ROL2(s[20], 18);
		s[20] = ROL2(s[2], 62);
		s[2] = ROL2(s[12], 43);
		s[12] = ROL2(s[13], 25);
		s[13] = ROL2(s[19], 8);
		s[19] = ROL2(s[23], 56);
		s[23] = ROL2(s[15], 41);
		s[15] = ROL2(s[4], 27);
		s[4] = ROL2(s[24], 14);
		s[24] = ROL2(s[21], 2);
		s[21] = ROL2(s[8], 55);
		s[8] = ROL2(s[16], 45);
		s[16] = ROL2(s[5], 36);
		s[5] = ROL2(s[3], 28);
		s[3] = ROL2(s[18], 21);
		s[18] = ROL2(s[17], 15);
		s[17] = ROL2(s[11], 10);
		s[11] = ROL2(s[7], 6);
		s[7] = ROL2(s[10], 3);
		s[10] = ROL2(v, 1);

		/* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
		v = s[0]; w = s[1]; s[0] ^= (~w) & s[2]; s[1] ^= (~s[2]) & s[3]; s[2] ^= (~s[3]) & s[4]; s[3] ^= (~s[4]) & v; s[4] ^= (~v) & w;
		v = s[5]; w = s[6]; s[5] ^= (~w) & s[7]; s[6] ^= (~s[7]) & s[8]; s[7] ^= (~s[8]) & s[9]; s[8] ^= (~s[9]) & v; s[9] ^= (~v) & w;
		v = s[10]; w = s[11]; s[10] ^= (~w) & s[12]; s[11] ^= (~s[12]) & s[13]; s[12] ^= (~s[13]) & s[14]; s[13] ^= (~s[14]) & v; s[14] ^= (~v) & w;
		v = s[15]; w = s[16]; s[15] ^= (~w) & s[17]; s[16] ^= (~s[17]) & s[18]; s[17] ^= (~s[18]) & s[19]; s[18] ^= (~s[19]) & v; s[19] ^= (~v) & w;
		v = s[20]; w = s[21]; s[20] ^= (~w) & s[22]; s[21] ^= (~s[22]) & s[23]; s[22] ^= (~s[23]) & s[24]; s[23] ^= (~s[24]) & v; s[24] ^= (~v) & w;

		/* iota: a[0,0] ^= round constant */
		s[0] ^= c_keccak_round_constants35[i];
	}
	t[0] = s[0] ^ s[5] ^ s[10] ^ s[15] ^ s[20];
	t[1] = s[1] ^ s[6] ^ s[11] ^ s[16] ^ s[21];
	t[2] = s[2] ^ s[7] ^ s[12] ^ s[17] ^ s[22];
	t[3] = s[3] ^ s[8] ^ s[13] ^ s[18] ^ s[23];
	t[4] = s[4] ^ s[9] ^ s[14] ^ s[19] ^ s[24];

	s[0] ^= t[4] ^ ROL2(t[1], 1);
	s[18] ^= t[2] ^ ROL2(t[4], 1);
	s[24] ^= t[3] ^ ROL2(t[0], 1);

	s[3] = ROL2(s[18], 21) ^ ((~ROL2(s[24], 14)) & s[0]);
}

__global__  __launch_bounds__(256, 2)
void quark_keccak512_gpu_hash_64(uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{
    uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

        int hashPosition = nounce - startNounce;
        uint64_t *inpHash = &g_hash[8 * hashPosition];

        uint2 keccak_gpu_state[25];
#pragma unroll
		for (int i = 0; i<8; i++)
		{
			keccak_gpu_state[i] = vectorize(inpHash[i]);
		}
		keccak_gpu_state[8] = make_uint2(0x00000001UL, 0x80000000);	//vectorize(0x8000000000000001ULL);

#pragma unroll
        for (int i=9; i<25; i++)
		{
			keccak_gpu_state[i] = make_uint2(0, 0);
		}
		keccak_block_35(keccak_gpu_state);

#pragma unroll
        for(int i=0;i<8;i++)
			inpHash[i] = devectorize(keccak_gpu_state[i]);
    }
}

__global__  __launch_bounds__(256, 2)
void quark_keccak512_gpu_hash_64_final(uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

		int hashPosition = nounce - startNounce;
		uint64_t *inpHash = &g_hash[8 * hashPosition];

		uint2 keccak_gpu_state[25];
#pragma unroll
		for (int i = 0; i<8; i++)
		{
			keccak_gpu_state[i] = vectorize(inpHash[i]);
		}
		keccak_gpu_state[8] = make_uint2(0x00000001UL, 0x80000000);	//vectorize(0x8000000000000001ULL);

#pragma unroll
		for (int i = 9; i<25; i++)
		{
			keccak_gpu_state[i] = make_uint2(0, 0);
		}
		keccak_block_35_final(keccak_gpu_state);

		inpHash[3] = devectorize(keccak_gpu_state[3]);
	}
}

__host__ void quark_keccak512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
    const uint32_t threadsperblock = 32;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

    quark_keccak512_gpu_hash_64<<<grid, block>>>(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);
}

__host__ void quark_keccak512_cpu_hash_64_final(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
	const uint32_t threadsperblock = 32;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	quark_keccak512_gpu_hash_64_final << <grid, block >> >(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);
}
