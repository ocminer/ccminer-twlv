#include "hip/hip_runtime.h"
// Auf QuarkCoin spezialisierte Version von Groestl inkl. Bitslice

#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"

#define TPB 256
#define THF 4

// aus cpu-miner.c
//extern short device_map[8];


// diese Struktur wird in der Init Funktion angefordert
//static hipDeviceProp_t props[8];

// 64 Register Variante f�r Compute 3.0
#include "groestl_functions_quad.cu"
#include "bitslice_transformations_quad.cu"

__global__ __launch_bounds__(TPB, THF)
void quark_groestl512_gpu_hash_64_quad(uint32_t threads, uint32_t startNounce, uint32_t *const __restrict__ g_hash, const uint32_t *const __restrict__ g_nonceVector)
{
	uint32_t msgBitsliced[8];
	uint32_t state[8];
	uint32_t hash[16];
	// durch 4 dividieren, weil jeweils 4 Threads zusammen ein Hash berechnen
    uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 2;
    if (thread < threads)
    {
        // GROESTL
        uint32_t nounce = g_nonceVector ? g_nonceVector[thread] : (startNounce + thread);
		uint32_t hashPosition = nounce - startNounce;
        uint32_t *inpHash = &g_hash[hashPosition * 16];

        const uint32_t thr = threadIdx.x & (THF-1);

		uint32_t message[8] =
		{
			inpHash[thr], inpHash[(THF)+thr], inpHash[(2 * THF) + thr], inpHash[(3 * THF) + thr],0, 0, 0, 
		};
		if (thr == 0) message[4] = 0x80UL;
		if (thr == 3) message[7] = 0x01000000UL;

		to_bitslice_quad(message, msgBitsliced);

        groestl512_progressMessage_quad(state, msgBitsliced);

		from_bitslice_quad(state, hash);
		if (thr == 0)
		{
			#pragma unroll
			for (int k = 0; k < 16; k++) inpHash[k] = hash[k];
		}
    }
}

__global__ void __launch_bounds__(TPB, THF)
quark_doublegroestl512_gpu_hash_64_quad(uint32_t threads, uint32_t startNounce, uint32_t * __restrict__ g_hash, uint32_t * __restrict__ g_nonceVector)
{
    uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x)>>2;
    if (thread < threads)
    {
        // GROESTL
        uint32_t message[8];
        uint32_t state[8];

        uint32_t nounce = g_nonceVector ? g_nonceVector[thread] : (startNounce + thread);

        int hashPosition = nounce - startNounce;
        uint32_t * inpHash = &g_hash[hashPosition<<4];
        const uint16_t thr = threadIdx.x & (THF-1);

        #pragma unroll
        for(int k=0;k<4;k++) message[k] = inpHash[(k * THF) + thr];

        #pragma unroll
        for(int k=4;k<8;k++) message[k] = 0;

        if (thr == 0) message[4] = 0x80;
        if (thr == 3) message[7] = 0x01000000;

        uint32_t msgBitsliced[8];
        to_bitslice_quad(message, msgBitsliced);

        for (int round=0; round<2; round++)
        {
            groestl512_progressMessage_quad(state, msgBitsliced);

            if (round < 1)
            {
                // Verkettung zweier Runden inclusive Padding.
                msgBitsliced[ 0] = __byte_perm(state[ 0], 0x00800100, 0x4341 + (((threadIdx.x&3)==3)<<13));
                msgBitsliced[ 1] = __byte_perm(state[ 1], 0x00800100, 0x4341);
                msgBitsliced[ 2] = __byte_perm(state[ 2], 0x00800100, 0x4341);
                msgBitsliced[ 3] = __byte_perm(state[ 3], 0x00800100, 0x4341);
                msgBitsliced[ 4] = __byte_perm(state[ 4], 0x00800100, 0x4341);
                msgBitsliced[ 5] = __byte_perm(state[ 5], 0x00800100, 0x4341);
                msgBitsliced[ 6] = __byte_perm(state[ 6], 0x00800100, 0x4341);
                msgBitsliced[ 7] = __byte_perm(state[ 7], 0x00800100, 0x4341 + (((threadIdx.x&3)==0)<<4));
            }
        }

        // Nur der erste von jeweils 4 Threads bekommt das Ergebns-Hash
        uint32_t *outpHash = inpHash;
        uint32_t hash[16];
        from_bitslice_quad(state, hash);

		if (thr != 0) return;

		#pragma unroll
        for(int k=0;k<16;k++) outpHash[k] = hash[k];
    }
}

// Setup-Funktionen
__host__ void quark_groestl512_cpu_init(int thr_id, uint32_t threads)
{
//    hipGetDeviceProperties(&props[thr_id], device_map[thr_id]);
}

__host__ void quark_groestl512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
    // Compute 3.0 benutzt die registeroptimierte Quad Variante mit Warp Shuffle
    // mit den Quad Funktionen brauchen wir jetzt 4 threads pro Hash, daher Faktor 4 bei der Blockzahl
    const int factor = THF;

    // berechne wie viele Thread Blocks wir brauchen
	dim3 grid(factor*((threads + TPB - 1) / TPB));
	dim3 block(TPB);

    quark_groestl512_gpu_hash_64_quad<<<grid, block>>>(threads, startNounce, d_hash, d_nonceVector);

    // Strategisches Sleep Kommando zur Senkung der CPU Last
	//MyStreamSynchronize(NULL, order, thr_id);
}

__host__ void quark_doublegroestl512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
    // Compute 3.0 benutzt die registeroptimierte Quad Variante mit Warp Shuffle
    // mit den Quad Funktionen brauchen wir jetzt 4 threads pro Hash, daher Faktor 4 bei der Blockzahl
    const int factor = THF;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid(factor*((threads + TPB-1)/TPB));
    dim3 block(TPB);

    quark_doublegroestl512_gpu_hash_64_quad<<<grid, block>>>(threads, startNounce, d_hash, d_nonceVector);
}
