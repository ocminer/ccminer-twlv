#include "hip/hip_runtime.h"
// Original version written by Schleicher (KlausT @github)
// Redistribution and use in source and binary forms, with or without modification, are permitted

#include <stdint.h>
#include "miner.h"
#include "cuda_helper.h"

void bitcoin_cpu_init(int thr_id);
void bitcoin_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce, const uint32_t *const ms, uint32_t merkle, uint32_t time, uint32_t compacttarget, uint32_t *const h_nounce);
void bitcoin_midstate(const uint32_t *data, uint32_t *midstate);

__constant__ uint32_t pTarget[8];
static uint32_t *d_result[MAX_GPUS];

#define TPB 512
#define NONCES_PER_THREAD 32

#define rrot(x, n) ((x >> n) | (x << (32 - n)))

__global__ __launch_bounds__(TPB, 2)
void bitcoin_gpu_hash(const uint32_t threads, const uint32_t startNounce, uint32_t *const result, const uint32_t t1c, const uint32_t t2c, const uint32_t w16, const uint32_t w16rot, const uint32_t w17, const uint32_t w17rot, const uint32_t b2, const uint32_t c2, const uint32_t d2, const uint32_t f2, const uint32_t g2, const uint32_t h2, const uint32_t ms0, const uint32_t ms1, const uint32_t ms2, const uint32_t ms3, const uint32_t ms4, const uint32_t ms5, const uint32_t ms6, const uint32_t ms7, const uint32_t compacttarget)
{
	uint32_t threadindex = (blockDim.x * blockIdx.x + threadIdx.x);
	if (threadindex < threads)
	{
		uint32_t t1, a, b, c, d, e, f, g, h;
		uint32_t w[64];
		const uint32_t numberofthreads = blockDim.x*gridDim.x;
		const uint32_t maxnonce = startNounce + threadindex + numberofthreads*NONCES_PER_THREAD - 1;
		const uint32_t threadindex = blockIdx.x*blockDim.x + threadIdx.x;
		

		#pragma unroll 
		for (uint32_t nonce = startNounce + threadindex; nonce <= maxnonce; nonce += numberofthreads)
		{
			w[18] = (rrot(nonce, 7) ^ rrot(nonce, 18) ^ (nonce >> 3)) + w16rot;
			w[19] = nonce + w17rot;
			w[20] = 0x80000000U + (rrot(w[18], 17) ^ rrot(w[18], 19) ^ (w[18] >> 10));
			w[21] = (rrot(w[19], 17) ^ rrot(w[19], 19) ^ (w[19] >> 10));
			w[22] = 0x280U + (rrot(w[20], 17) ^ rrot(w[20], 19) ^ (w[20] >> 10));
			w[23] = w16 + (rrot(w[21], 17) ^ rrot(w[21], 19) ^ (w[21] >> 10));
			w[24] = w17 + (rrot(w[22], 17) ^ rrot(w[22], 19) ^ (w[22] >> 10));
			w[25] = w[18] + (rrot(w[23], 17) ^ rrot(w[23], 19) ^ (w[23] >> 10));
			w[26] = w[19] + (rrot(w[24], 17) ^ rrot(w[24], 19) ^ (w[24] >> 10));
			w[27] = w[20] + (rrot(w[25], 17) ^ rrot(w[25], 19) ^ (w[25] >> 10));
			w[28] = w[21] + (rrot(w[26], 17) ^ rrot(w[26], 19) ^ (w[26] >> 10));
			w[29] = w[22] + (rrot(w[27], 17) ^ rrot(w[27], 19) ^ (w[27] >> 10));
			w[30] = w[23] + 0xa00055U + (rrot(w[28], 17) ^ rrot(w[28], 19) ^ (w[28] >> 10));
			w[31] = 0x280U + w[24] + (rrot(w16, 7) ^ rrot(w16, 18) ^ (w16 >> 3)) + (rrot(w[29], 17) ^ rrot(w[29], 19) ^ (w[29] >> 10));
			w[32] = w16 + w[25] + (rrot(w17, 7) ^ rrot(w17, 18) ^ (w17 >> 3)) + (rrot(w[30], 17) ^ rrot(w[30], 19) ^ (w[30] >> 10));
			w[33] = w17 + w[26] + (rrot(w[18], 7) ^ rrot(w[18], 18) ^ (w[18] >> 3)) + (rrot(w[31], 17) ^ rrot(w[31], 19) ^ (w[31] >> 10));
#pragma unroll
			for (int i = 34; i < 62; i++)
				w[i] = w[i-16] + w[i-7] + (rrot(w[i-15], 7) ^ rrot(w[i-15], 18) ^ (w[i-15] >> 3)) + (rrot(w[i-2], 17) ^ rrot(w[i-2], 19) ^ (w[i-2] >> 10));

			t1 = t1c + (uint32_t)nonce;
			a = ms0 + t1;
			e = t1 + t2c;
			//
			t1 = d2 + (rrot(a, 6) ^ rrot(a, 11) ^ rrot(a, 25)) + (c2 ^ (a & (b2 ^ c2))) + 0xb956c25bU;
			h = h2 + t1;
			d = t1 + (rrot(e, 2) ^ rrot(e, 13) ^ rrot(e, 22)) + ((g2 & f2) | (e & (g2 | f2)));
			//
			t1 = c2 + (rrot(h, 6) ^ rrot(h, 11) ^ rrot(h, 25)) + (b2 ^ (h & (a ^ b2))) + 0x59f111f1U;
			g = g2 + t1;
			c = t1 + (rrot(d, 2) ^ rrot(d, 13) ^ rrot(d, 22)) + ((f2 & e) | (d & (f2 | e)));
			//
			t1 = b2 + (rrot(g, 6) ^ rrot(g, 11) ^ rrot(g, 25)) + (a ^ (g & (h ^ a))) + 0x923f82a4U;
			f = f2 + t1;
			b = t1 + (rrot(c, 2) ^ rrot(c, 13) ^ rrot(c, 22)) + ((e & d) | (c & (e | d)));
			//
			t1 = a + (rrot(f, 6) ^ rrot(f, 11) ^ rrot(f, 25)) + (h ^ (f & (g ^ h))) + 0xab1c5ed5U;
			e += t1;
			a = t1 + (rrot(b, 2) ^ rrot(b, 13) ^ rrot(b, 22)) + ((d & c) | (b & (d | c)));
			//
			t1 = h + (rrot(e, 6) ^ rrot(e, 11) ^ rrot(e, 25)) + (g ^ (e & (f ^ g))) + 0xd807aa98U;
			d += t1;
			h = t1 + (rrot(a, 2) ^ rrot(a, 13) ^ rrot(a, 22)) + ((c & b) | (a & (c | b)));
			//
			t1 = g + (rrot(d, 6) ^ rrot(d, 11) ^ rrot(d, 25)) + (f ^ (d & (e ^ f))) + 0x12835b01U;
			c += t1;
			g = t1 + (rrot(h, 2) ^ rrot(h, 13) ^ rrot(h, 22)) + ((b & a) | (h & (b | a)));
			//
			t1 = f + (rrot(c, 6) ^ rrot(c, 11) ^ rrot(c, 25)) + (e ^ (c & (d ^ e))) + 0x243185beU;
			b += t1;
			f = t1 + (rrot(g, 2) ^ rrot(g, 13) ^ rrot(g, 22)) + ((a & h) | (g & (a | h)));
			//
			t1 = e + (rrot(b, 6) ^ rrot(b, 11) ^ rrot(b, 25)) + (d ^ (b & (c ^ d))) + 0x550c7dc3U;
			a += t1;
			e = t1 + (rrot(f, 2) ^ rrot(f, 13) ^ rrot(f, 22)) + ((h & g) | (f & (h | g)));
			//
			t1 = d + (rrot(a, 6) ^ rrot(a, 11) ^ rrot(a, 25)) + (c ^ (a & (b ^ c))) + 0x72be5d74U;
			h += t1;
			d = t1 + (rrot(e, 2) ^ rrot(e, 13) ^ rrot(e, 22)) + ((g & f) | (e & (g | f)));
			//
			t1 = c + (rrot(h, 6) ^ rrot(h, 11) ^ rrot(h, 25)) + (b ^ (h & (a ^ b))) + 0x80deb1feU;
			g += t1;
			c = t1 + (rrot(d, 2) ^ rrot(d, 13) ^ rrot(d, 22)) + ((f & e) | (d & (f | e)));
			//
			t1 = b + (rrot(g, 6) ^ rrot(g, 11) ^ rrot(g, 25)) + (a ^ (g & (h ^ a))) + 0x9bdc06a7U;
			f += t1;
			b = t1 + (rrot(c, 2) ^ rrot(c, 13) ^ rrot(c, 22)) + ((e & d) | (c & (e | d)));
			//
			t1 = a + (rrot(f, 6) ^ rrot(f, 11) ^ rrot(f, 25)) + (h ^ (f & (g ^ h))) + 0xc19bf3f4U;
			e += t1;
			a = t1 + (rrot(b, 2) ^ rrot(b, 13) ^ rrot(b, 22)) + ((d & c) | (b & (d | c)));
			//
			t1 = h + (rrot(e, 6) ^ rrot(e, 11) ^ rrot(e, 25)) + (g ^ (e & (f ^ g))) + 0xe49b69c1U + w16;
			d += t1;
			h = t1 + (rrot(a, 2) ^ rrot(a, 13) ^ rrot(a, 22)) + ((c & b) | (a & (c | b)));
			//
			t1 = g + (rrot(d, 6) ^ rrot(d, 11) ^ rrot(d, 25)) + (f ^ (d & (e ^ f))) + 0xefbe4786U + w17;
			c += t1;
			g = t1 + (rrot(h, 2) ^ rrot(h, 13) ^ rrot(h, 22)) + ((b & a) | (h & (b | a)));
			//
			t1 = f + (rrot(c, 6) ^ rrot(c, 11) ^ rrot(c, 25)) + (e ^ (c & (d ^ e))) + 0x0fc19dc6U + w[18];
			b += t1;
			f = t1 + (rrot(g, 2) ^ rrot(g, 13) ^ rrot(g, 22)) + ((a & h) | (g & (a | h)));
			//
			t1 = e + (rrot(b, 6) ^ rrot(b, 11) ^ rrot(b, 25)) + (d ^ (b & (c ^ d))) + 0x240ca1ccU + w[19];
			a += t1;
			e = t1 + (rrot(f, 2) ^ rrot(f, 13) ^ rrot(f, 22)) + ((h & g) | (f & (h | g)));
			//
			t1 = d + (rrot(a, 6) ^ rrot(a, 11) ^ rrot(a, 25)) + (c ^ (a & (b ^ c))) + 0x2de92c6fU + w[20];
			h += t1;
			d = t1 + (rrot(e, 2) ^ rrot(e, 13) ^ rrot(e, 22)) + ((g & f) | (e & (g | f)));
			//
			t1 = c + (rrot(h, 6) ^ rrot(h, 11) ^ rrot(h, 25)) + (b ^ (h & (a ^ b))) + 0x4a7484aaU + w[21];
			g += t1;
			c = t1 + (rrot(d, 2) ^ rrot(d, 13) ^ rrot(d, 22)) + ((f & e) | (d & (f | e)));
			//
			t1 = b + (rrot(g, 6) ^ rrot(g, 11) ^ rrot(g, 25)) + (a ^ (g & (h ^ a))) + 0x5cb0a9dcU + w[22];
			f += t1;
			b = t1 + (rrot(c, 2) ^ rrot(c, 13) ^ rrot(c, 22)) + ((e & d) | (c & (e | d)));
			//
			t1 = a + (rrot(f, 6) ^ rrot(f, 11) ^ rrot(f, 25)) + (h ^ (f & (g ^ h))) + 0x76f988daU + w[23];
			e += t1;
			a = t1 + (rrot(b, 2) ^ rrot(b, 13) ^ rrot(b, 22)) + ((d & c) | (b & (d | c)));
			//
			t1 = h + (rrot(e, 6) ^ rrot(e, 11) ^ rrot(e, 25)) + (g ^ (e & (f ^ g))) + 0x983e5152U + w[24];
			d += t1;
			h = t1 + (rrot(a, 2) ^ rrot(a, 13) ^ rrot(a, 22)) + ((c & b) | (a & (c | b)));
			//
			t1 = g + (rrot(d, 6) ^ rrot(d, 11) ^ rrot(d, 25)) + (f ^ (d & (e ^ f))) + 0xa831c66dU + w[25];
			c += t1;
			g = t1 + (rrot(h, 2) ^ rrot(h, 13) ^ rrot(h, 22)) + ((b & a) | (h & (b | a)));
			//
			t1 = f + (rrot(c, 6) ^ rrot(c, 11) ^ rrot(c, 25)) + (e ^ (c & (d ^ e))) + 0xb00327c8U + w[26];
			b += t1;
			f = t1 + (rrot(g, 2) ^ rrot(g, 13) ^ rrot(g, 22)) + ((a & h) | (g & (a | h)));
			//
			t1 = e + (rrot(b, 6) ^ rrot(b, 11) ^ rrot(b, 25)) + (d ^ (b & (c ^ d))) + 0xbf597fc7U + w[27];
			a += t1;
			e = t1 + (rrot(f, 2) ^ rrot(f, 13) ^ rrot(f, 22)) + ((h & g) | (f & (h | g)));
			//
			t1 = d + (rrot(a, 6) ^ rrot(a, 11) ^ rrot(a, 25)) + (c ^ (a & (b ^ c))) + 0xc6e00bf3U + w[28];
			h += t1;
			d = t1 + (rrot(e, 2) ^ rrot(e, 13) ^ rrot(e, 22)) + ((g & f) | (e & (g | f)));
			//
			t1 = c + (rrot(h, 6) ^ rrot(h, 11) ^ rrot(h, 25)) + (b ^ (h & (a ^ b))) + 0xd5a79147U + w[29];
			g += t1;
			c = t1 + (rrot(d, 2) ^ rrot(d, 13) ^ rrot(d, 22)) + ((f & e) | (d & (f | e)));
			//
			t1 = b + (rrot(g, 6) ^ rrot(g, 11) ^ rrot(g, 25)) + (a ^ (g & (h ^ a))) + 0x06ca6351U + w[30];
			f += t1;
			b = t1 + (rrot(c, 2) ^ rrot(c, 13) ^ rrot(c, 22)) + ((e & d) | (c & (e | d)));
			//
			t1 = a + (rrot(f, 6) ^ rrot(f, 11) ^ rrot(f, 25)) + (h ^ (f & (g ^ h))) + 0x14292967U + w[31];
			e += t1;
			a = t1 + (rrot(b, 2) ^ rrot(b, 13) ^ rrot(b, 22)) + ((d & c) | (b & (d | c)));
			//
			t1 = h + (rrot(e, 6) ^ rrot(e, 11) ^ rrot(e, 25)) + (g ^ (e & (f ^ g))) + 0x27b70a85U + w[32];
			d += t1;
			h = t1 + (rrot(a, 2) ^ rrot(a, 13) ^ rrot(a, 22)) + ((c & b) | (a & (c | b)));
			//
			t1 = g + (rrot(d, 6) ^ rrot(d, 11) ^ rrot(d, 25)) + (f ^ (d & (e ^ f))) + 0x2e1b2138U + w[33];
			c += t1;
			g = t1 + (rrot(h, 2) ^ rrot(h, 13) ^ rrot(h, 22)) + ((b & a) | (h & (b | a)));
			//
			t1 = f + (rrot(c, 6) ^ rrot(c, 11) ^ rrot(c, 25)) + (e ^ (c & (d ^ e))) + 0x4d2c6dfcU + w[34];
			b += t1;
			f = t1 + (rrot(g, 2) ^ rrot(g, 13) ^ rrot(g, 22)) + ((a & h) | (g & (a | h)));
			//
			t1 = e + (rrot(b, 6) ^ rrot(b, 11) ^ rrot(b, 25)) + (d ^ (b & (c ^ d))) + 0x53380d13U + w[35];
			a += t1;
			e = t1 + (rrot(f, 2) ^ rrot(f, 13) ^ rrot(f, 22)) + ((h & g) | (f & (h | g)));
			//
			t1 = d + (rrot(a, 6) ^ rrot(a, 11) ^ rrot(a, 25)) + (c ^ (a & (b ^ c))) + 0x650a7354U + w[36];
			h += t1;
			d = t1 + (rrot(e, 2) ^ rrot(e, 13) ^ rrot(e, 22)) + ((g & f) | (e & (g | f)));
			//
			t1 = c + (rrot(h, 6) ^ rrot(h, 11) ^ rrot(h, 25)) + (b ^ (h & (a ^ b))) + 0x766a0abbU + w[37];
			g += t1;
			c = t1 + (rrot(d, 2) ^ rrot(d, 13) ^ rrot(d, 22)) + ((f & e) | (d & (f | e)));
			//
			t1 = b + (rrot(g, 6) ^ rrot(g, 11) ^ rrot(g, 25)) + (a ^ (g & (h ^ a))) + 0x81c2c92eU + w[38];
			f += t1;
			b = t1 + (rrot(c, 2) ^ rrot(c, 13) ^ rrot(c, 22)) + ((e & d) | (c & (e | d)));
			//
			t1 = a + (rrot(f, 6) ^ rrot(f, 11) ^ rrot(f, 25)) + (h ^ (f & (g ^ h))) + 0x92722c85U + w[39];
			e += t1;
			a = t1 + (rrot(b, 2) ^ rrot(b, 13) ^ rrot(b, 22)) + ((d & c) | (b & (d | c)));
			//
			t1 = h + (rrot(e, 6) ^ rrot(e, 11) ^ rrot(e, 25)) + (g ^ (e & (f ^ g))) + 0xa2bfe8a1U + w[40];
			d += t1;
			h = t1 + (rrot(a, 2) ^ rrot(a, 13) ^ rrot(a, 22)) + ((c & b) | (a & (c | b)));
			//
			t1 = g + (rrot(d, 6) ^ rrot(d, 11) ^ rrot(d, 25)) + (f ^ (d & (e ^ f))) + 0xa81a664bU + w[41];
			c += t1;
			g = t1 + (rrot(h, 2) ^ rrot(h, 13) ^ rrot(h, 22)) + ((b & a) | (h & (b | a)));
			//
			t1 = f + (rrot(c, 6) ^ rrot(c, 11) ^ rrot(c, 25)) + (e ^ (c & (d ^ e))) + 0xc24b8b70U + w[42];
			b += t1;
			f = t1 + (rrot(g, 2) ^ rrot(g, 13) ^ rrot(g, 22)) + ((a & h) | (g & (a | h)));
			//
			t1 = e + (rrot(b, 6) ^ rrot(b, 11) ^ rrot(b, 25)) + (d ^ (b & (c ^ d))) + 0xc76c51a3U + w[43];
			a += t1;
			e = t1 + (rrot(f, 2) ^ rrot(f, 13) ^ rrot(f, 22)) + ((h & g) | (f & (h | g)));
			//
			t1 = d + (rrot(a, 6) ^ rrot(a, 11) ^ rrot(a, 25)) + (c ^ (a & (b ^ c))) + 0xd192e819U + w[44];
			h += t1;
			d = t1 + (rrot(e, 2) ^ rrot(e, 13) ^ rrot(e, 22)) + ((g & f) | (e & (g | f)));
			//
			t1 = c + (rrot(h, 6) ^ rrot(h, 11) ^ rrot(h, 25)) + (b ^ (h & (a ^ b))) + 0xd6990624U + w[45];
			g += t1;
			c = t1 + (rrot(d, 2) ^ rrot(d, 13) ^ rrot(d, 22)) + ((f & e) | (d & (f | e)));
			//
			t1 = b + (rrot(g, 6) ^ rrot(g, 11) ^ rrot(g, 25)) + (a ^ (g & (h ^ a))) + 0xf40e3585U + w[46];
			f += t1;
			b = t1 + (rrot(c, 2) ^ rrot(c, 13) ^ rrot(c, 22)) + ((e & d) | (c & (e | d)));
			//
			t1 = a + (rrot(f, 6) ^ rrot(f, 11) ^ rrot(f, 25)) + (h ^ (f & (g ^ h))) + 0x106aa070U + w[47];
			e += t1;
			a = t1 + (rrot(b, 2) ^ rrot(b, 13) ^ rrot(b, 22)) + ((d & c) | (b & (d | c)));
			//
			t1 = h + (rrot(e, 6) ^ rrot(e, 11) ^ rrot(e, 25)) + (g ^ (e & (f ^ g))) + 0x19a4c116U + w[48];
			d += t1;
			h = t1 + (rrot(a, 2) ^ rrot(a, 13) ^ rrot(a, 22)) + ((c & b) | (a & (c | b)));
			//
			t1 = g + (rrot(d, 6) ^ rrot(d, 11) ^ rrot(d, 25)) + (f ^ (d & (e ^ f))) + 0x1e376c08U + w[49];
			c += t1;
			g = t1 + (rrot(h, 2) ^ rrot(h, 13) ^ rrot(h, 22)) + ((b & a) | (h & (b | a)));
			//
			t1 = f + (rrot(c, 6) ^ rrot(c, 11) ^ rrot(c, 25)) + (e ^ (c & (d ^ e))) + 0x2748774cU + w[50];
			b += t1;
			f = t1 + (rrot(g, 2) ^ rrot(g, 13) ^ rrot(g, 22)) + ((a & h) | (g & (a | h)));
			//
			t1 = e + (rrot(b, 6) ^ rrot(b, 11) ^ rrot(b, 25)) + (d ^ (b & (c ^ d))) + 0x34b0bcb5U + w[51];
			a += t1;
			e = t1 + (rrot(f, 2) ^ rrot(f, 13) ^ rrot(f, 22)) + ((h & g) | (f & (h | g)));
			//
			t1 = d + (rrot(a, 6) ^ rrot(a, 11) ^ rrot(a, 25)) + (c ^ (a & (b ^ c))) + 0x391c0cb3U + w[52];
			h += t1;
			d = t1 + (rrot(e, 2) ^ rrot(e, 13) ^ rrot(e, 22)) + ((g & f) | (e & (g | f)));
			//
			t1 = c + (rrot(h, 6) ^ rrot(h, 11) ^ rrot(h, 25)) + (b ^ (h & (a ^ b))) + 0x4ed8aa4aU + w[53];
			g += t1;
			c = t1 + (rrot(d, 2) ^ rrot(d, 13) ^ rrot(d, 22)) + ((f & e) | (d & (f | e)));
			//
			t1 = b + (rrot(g, 6) ^ rrot(g, 11) ^ rrot(g, 25)) + (a ^ (g & (h ^ a))) + 0x5b9cca4fU + w[54];
			f += t1;
			b = t1 + (rrot(c, 2) ^ rrot(c, 13) ^ rrot(c, 22)) + ((e & d) | (c & (e | d)));
			//
			t1 = a + (rrot(f, 6) ^ rrot(f, 11) ^ rrot(f, 25)) + (h ^ (f & (g ^ h))) + 0x682e6ff3U + w[55];
			e += t1;
			a = t1 + (rrot(b, 2) ^ rrot(b, 13) ^ rrot(b, 22)) + ((d & c) | (b & (d | c)));
			//
			t1 = h + (rrot(e, 6) ^ rrot(e, 11) ^ rrot(e, 25)) + (g ^ (e & (f ^ g))) + 0x748f82eeU + w[56];
			d += t1;
			h = t1 + (rrot(a, 2) ^ rrot(a, 13) ^ rrot(a, 22)) + ((c & b) | (a & (c | b)));
			//
			t1 = g + (rrot(d, 6) ^ rrot(d, 11) ^ rrot(d, 25)) + (f ^ (d & (e ^ f))) + 0x78a5636fU + w[57];
			c += t1;
			g = t1 + (rrot(h, 2) ^ rrot(h, 13) ^ rrot(h, 22)) + ((b & a) | (h & (b | a)));
			//
			t1 = f + (rrot(c, 6) ^ rrot(c, 11) ^ rrot(c, 25)) + (e ^ (c & (d ^ e))) + 0x84c87814U + w[58];
			b += t1;
			f = t1 + (rrot(g, 2) ^ rrot(g, 13) ^ rrot(g, 22)) + ((a & h) | (g & (a | h)));
			//
			t1 = e + (rrot(b, 6) ^ rrot(b, 11) ^ rrot(b, 25)) + (d ^ (b & (c ^ d))) + 0x8cc70208U + w[59];
			a += t1;
			e = t1 + (rrot(f, 2) ^ rrot(f, 13) ^ rrot(f, 22)) + ((h & g) | (f & (h | g)));
			//
			t1 = d + (rrot(a, 6) ^ rrot(a, 11) ^ rrot(a, 25)) + (c ^ (a & (b ^ c))) + 0x90befffaU + w[60];
			h += t1;
			d = t1 + (rrot(e, 2) ^ rrot(e, 13) ^ rrot(e, 22)) + ((g & f) | (e & (g | f)));
			//
			t1 = c + (rrot(h, 6) ^ rrot(h, 11) ^ rrot(h, 25)) + (b ^ (h & (a ^ b))) + 0xa4506cebU + w[61];
			g += t1;
			c = t1 + (rrot(d, 2) ^ rrot(d, 13) ^ rrot(d, 22)) + ((f & e) | (d & (f | e)));
			//
			t1 = b + (rrot(g, 6) ^ rrot(g, 11) ^ rrot(g, 25)) + (a ^ (g & (h ^ a))) + 0xbef9a3f7U + w[46] + w[55] + (rrot(w[47], 7) ^ rrot(w[47], 18) ^ (w[47] >> 3)) + (rrot(w[60], 17) ^ rrot(w[60], 19) ^ (w[60] >> 10));
			f += t1;
			b = t1 + (rrot(c, 2) ^ rrot(c, 13) ^ rrot(c, 22)) + ((e & d) | (c & (e | d)));
			//
			t1 = a + (rrot(f, 6) ^ rrot(f, 11) ^ rrot(f, 25)) + (h ^ (f & (g ^ h))) + 0xc67178f2U + w[47] + w[56] + (rrot(w[48], 7) ^ rrot(w[48], 18) ^ (w[48] >> 3)) + (rrot(w[61], 17) ^ rrot(w[61], 19) ^ (w[61] >> 10));
			e += t1;
			a = t1 + (rrot(b, 2) ^ rrot(b, 13) ^ rrot(b, 22)) + ((d & c) | (b & (d | c)));
			//
			w[0] = a + ms0; w[1] = b + ms1; w[2] = c + ms2; w[3] = d + ms3;
			w[4] = e + ms4; w[5] = f + ms5; w[6] = g + ms6; w[7] = h + ms7;
			// hash the hash ***************************************************************
			w[16] = w[0] + (rrot(w[1], 7) ^ rrot(w[1], 18) ^ (w[1] >> 3));
			w[17] = w[1] + (rrot(w[2], 7) ^ rrot(w[2], 18) ^ (w[2] >> 3)) + (rrot(0x100, 17) ^ rrot(0x100, 19) ^ (0x100 >> 10));
			w[18] = w[2] + (rrot(w[3], 7) ^ rrot(w[3], 18) ^ (w[3] >> 3)) + (rrot(w[16], 17) ^ rrot(w[16], 19) ^ (w[16] >> 10));
			w[19] = w[3] + (rrot(w[4], 7) ^ rrot(w[4], 18) ^ (w[4] >> 3)) + (rrot(w[17], 17) ^ rrot(w[17], 19) ^ (w[17] >> 10));
			w[20] = w[4] + (rrot(w[5], 7) ^ rrot(w[5], 18) ^ (w[5] >> 3)) + (rrot(w[18], 17) ^ rrot(w[18], 19) ^ (w[18] >> 10));
			w[21] = w[5] + (rrot(w[6], 7) ^ rrot(w[6], 18) ^ (w[6] >> 3)) + (rrot(w[19], 17) ^ rrot(w[19], 19) ^ (w[19] >> 10));
			w[22] = w[6] + 0x100 + (rrot(w[7], 7) ^ rrot(w[7], 18) ^ (w[7] >> 3)) + (rrot(w[20], 17) ^ rrot(w[20], 19) ^ (w[20] >> 10));
			w[23] = w[7] + w[16] + 0x11002000U + (rrot(w[21], 17) ^ rrot(w[21], 19) ^ (w[21] >> 10));
			w[24] = 0x80000000U + w[17] + (rrot(w[22], 17) ^ rrot(w[22], 19) ^ (w[22] >> 10));
			w[25] = w[18] + (rrot(w[23], 17) ^ rrot(w[23], 19) ^ (w[23] >> 10));
			w[26] = w[19] + (rrot(w[24], 17) ^ rrot(w[24], 19) ^ (w[24] >> 10));
			w[27] = w[20] + (rrot(w[25], 17) ^ rrot(w[25], 19) ^ (w[25] >> 10));
			w[28] = w[21] + (rrot(w[26], 17) ^ rrot(w[26], 19) ^ (w[26] >> 10));
			w[29] = w[22] + (rrot(w[27], 17) ^ rrot(w[27], 19) ^ (w[27] >> 10));
			w[30] = w[23] + (rrot(0x100, 7) ^ rrot(0x100, 18) ^ (0x100 >> 3)) + (rrot(w[28], 17) ^ rrot(w[28], 19) ^ (w[28] >> 10));
			w[31] = 0x100 + w[24] + (rrot(w[16], 7) ^ rrot(w[16], 18) ^ (w[16] >> 3)) + (rrot(w[29], 17) ^ rrot(w[29], 19) ^ (w[29] >> 10));
#pragma unroll
			for (int i = 32; i < 59; i++)
				w[i] = w[i - 16] + w[i - 7] + (rrot(w[i - 15], 7) ^ rrot(w[i - 15], 18) ^ (w[i - 15] >> 3)) + (rrot(w[i - 2], 17) ^ rrot(w[i - 2], 19) ^ (w[i - 2] >> 10));

			d = 0x98c7e2a2U + w[0];
			h = 0xfc08884dU + w[0];
			//
			t1 = (rrot(d, 6) ^ rrot(d, 11) ^ rrot(d, 25)) + (0x9b05688cU ^ (d & 0xca0b3af3)) + 0x90bb1e3cU + w[1];
			c = 0x3c6ef372U + t1;
			g = t1 + (rrot(h, 2) ^ rrot(h, 13) ^ rrot(h, 22)) + (0x2A01A605 | (h & 0xfb6feee7));
			//
			t1 = (rrot(c, 6) ^ rrot(c, 11) ^ rrot(c, 25)) + (0x510e527fU ^ (c & (d ^ 0x510e527fU))) + 0x50C6645BU + w[2];
			b = 0xbb67ae85U + t1;
			f = t1 + (rrot(g, 2) ^ rrot(g, 13) ^ rrot(g, 22)) + ((0x6a09e667U & h) | (g & (0x6a09e667U | h)));
			//
			t1 = (rrot(b, 6) ^ rrot(b, 11) ^ rrot(b, 25)) + (d ^ (b & (c ^ d))) + 0x3AC42E24U + w[3];
			a = 0x6a09e667U + t1;
			e = t1 + (rrot(f, 2) ^ rrot(f, 13) ^ rrot(f, 22)) + ((h & g) | (f & (h | g)));
			//
			t1 = d + (rrot(a, 6) ^ rrot(a, 11) ^ rrot(a, 25)) + (c ^ (a & (b ^ c))) + 0x3956c25bU + w[4];
			h += t1;
			d = t1 + (rrot(e, 2) ^ rrot(e, 13) ^ rrot(e, 22)) + ((g & f) | (e & (g | f)));
			//
			t1 = c + (rrot(h, 6) ^ rrot(h, 11) ^ rrot(h, 25)) + (b ^ (h & (a ^ b))) + 0x59f111f1U + w[5];
			g += t1;
			c = t1 + (rrot(d, 2) ^ rrot(d, 13) ^ rrot(d, 22)) + ((f & e) | (d & (f | e)));
			//
			t1 = b + (rrot(g, 6) ^ rrot(g, 11) ^ rrot(g, 25)) + (a ^ (g & (h ^ a))) + 0x923f82a4U + w[6];
			f += t1;
			b = t1 + (rrot(c, 2) ^ rrot(c, 13) ^ rrot(c, 22)) + ((e & d) | (c & (e | d)));
			//
			t1 = a + (rrot(f, 6) ^ rrot(f, 11) ^ rrot(f, 25)) + (h ^ (f & (g ^ h))) + 0xab1c5ed5U + w[7];
			e += t1;
			a = t1 + (rrot(b, 2) ^ rrot(b, 13) ^ rrot(b, 22)) + ((d & c) | (b & (d | c)));
			//
			t1 = h + (rrot(e, 6) ^ rrot(e, 11) ^ rrot(e, 25)) + (g ^ (e & (f ^ g))) + 0x5807aa98U;
			d += t1;
			h = t1 + (rrot(a, 2) ^ rrot(a, 13) ^ rrot(a, 22)) + ((c & b) | (a & (c | b)));
			//
			t1 = g + (rrot(d, 6) ^ rrot(d, 11) ^ rrot(d, 25)) + (f ^ (d & (e ^ f))) + 0x12835b01U;
			c += t1;
			g = t1 + (rrot(h, 2) ^ rrot(h, 13) ^ rrot(h, 22)) + ((b & a) | (h & (b | a)));
			//
			t1 = f + (rrot(c, 6) ^ rrot(c, 11) ^ rrot(c, 25)) + (e ^ (c & (d ^ e))) + 0x243185beU;
			b += t1;
			f = t1 + (rrot(g, 2) ^ rrot(g, 13) ^ rrot(g, 22)) + ((a & h) | (g & (a | h)));
			//
			t1 = e + (rrot(b, 6) ^ rrot(b, 11) ^ rrot(b, 25)) + (d ^ (b & (c ^ d))) + 0x550c7dc3U;
			a += t1;
			e = t1 + (rrot(f, 2) ^ rrot(f, 13) ^ rrot(f, 22)) + ((h & g) | (f & (h | g)));
			//
			t1 = d + (rrot(a, 6) ^ rrot(a, 11) ^ rrot(a, 25)) + (c ^ (a & (b ^ c))) + 0x72be5d74U;
			h += t1;
			d = t1 + (rrot(e, 2) ^ rrot(e, 13) ^ rrot(e, 22)) + ((g & f) | (e & (g | f)));
			//
			t1 = c + (rrot(h, 6) ^ rrot(h, 11) ^ rrot(h, 25)) + (b ^ (h & (a ^ b))) + 0x80deb1feU;
			g += t1;
			c = t1 + (rrot(d, 2) ^ rrot(d, 13) ^ rrot(d, 22)) + ((f & e) | (d & (f | e)));
			//
			t1 = b + (rrot(g, 6) ^ rrot(g, 11) ^ rrot(g, 25)) + (a ^ (g & (h ^ a))) + 0x9bdc06a7U;
			f += t1;
			b = t1 + (rrot(c, 2) ^ rrot(c, 13) ^ rrot(c, 22)) + ((e & d) | (c & (e | d)));
			//
			t1 = a + (rrot(f, 6) ^ rrot(f, 11) ^ rrot(f, 25)) + (h ^ (f & (g ^ h))) + 0xc19bf274U;
			e += t1;
			a = t1 + (rrot(b, 2) ^ rrot(b, 13) ^ rrot(b, 22)) + ((d & c) | (b & (d | c)));
			//
			t1 = h + (rrot(e, 6) ^ rrot(e, 11) ^ rrot(e, 25)) + (g ^ (e & (f ^ g))) + 0xe49b69c1U + w[16];
			d += t1;
			h = t1 + (rrot(a, 2) ^ rrot(a, 13) ^ rrot(a, 22)) + ((c & b) | (a & (c | b)));
			//
			t1 = g + (rrot(d, 6) ^ rrot(d, 11) ^ rrot(d, 25)) + (f ^ (d & (e ^ f))) + 0xefbe4786U + w[17];
			c += t1;
			g = t1 + (rrot(h, 2) ^ rrot(h, 13) ^ rrot(h, 22)) + ((b & a) | (h & (b | a)));
			//
			t1 = f + (rrot(c, 6) ^ rrot(c, 11) ^ rrot(c, 25)) + (e ^ (c & (d ^ e))) + 0x0fc19dc6U + w[18];
			b += t1;
			f = t1 + (rrot(g, 2) ^ rrot(g, 13) ^ rrot(g, 22)) + ((a & h) | (g & (a | h)));
			//
			t1 = e + (rrot(b, 6) ^ rrot(b, 11) ^ rrot(b, 25)) + (d ^ (b & (c ^ d))) + 0x240ca1ccU + w[19];
			a += t1;
			e = t1 + (rrot(f, 2) ^ rrot(f, 13) ^ rrot(f, 22)) + ((h & g) | (f & (h | g)));
			//
			t1 = d + (rrot(a, 6) ^ rrot(a, 11) ^ rrot(a, 25)) + (c ^ (a & (b ^ c))) + 0x2de92c6fU + w[20];
			h += t1;
			d = t1 + (rrot(e, 2) ^ rrot(e, 13) ^ rrot(e, 22)) + ((g & f) | (e & (g | f)));
			//
			t1 = c + (rrot(h, 6) ^ rrot(h, 11) ^ rrot(h, 25)) + (b ^ (h & (a ^ b))) + 0x4a7484aaU + w[21];
			g += t1;
			c = t1 + (rrot(d, 2) ^ rrot(d, 13) ^ rrot(d, 22)) + ((f & e) | (d & (f | e)));
			//
			t1 = b + (rrot(g, 6) ^ rrot(g, 11) ^ rrot(g, 25)) + (a ^ (g & (h ^ a))) + 0x5cb0a9dcU + w[22];
			f += t1;
			b = t1 + (rrot(c, 2) ^ rrot(c, 13) ^ rrot(c, 22)) + ((e & d) | (c & (e | d)));
			//
			t1 = a + (rrot(f, 6) ^ rrot(f, 11) ^ rrot(f, 25)) + (h ^ (f & (g ^ h))) + 0x76f988daU + w[23];
			e += t1;
			a = t1 + (rrot(b, 2) ^ rrot(b, 13) ^ rrot(b, 22)) + ((d & c) | (b & (d | c)));
			//
			t1 = h + (rrot(e, 6) ^ rrot(e, 11) ^ rrot(e, 25)) + (g ^ (e & (f ^ g))) + 0x983e5152U + w[24];
			d += t1;
			h = t1 + (rrot(a, 2) ^ rrot(a, 13) ^ rrot(a, 22)) + ((c & b) | (a & (c | b)));
			//
			t1 = g + (rrot(d, 6) ^ rrot(d, 11) ^ rrot(d, 25)) + (f ^ (d & (e ^ f))) + 0xa831c66dU + w[25];
			c += t1;
			g = t1 + (rrot(h, 2) ^ rrot(h, 13) ^ rrot(h, 22)) + ((b & a) | (h & (b | a)));
			//
			t1 = f + (rrot(c, 6) ^ rrot(c, 11) ^ rrot(c, 25)) + (e ^ (c & (d ^ e))) + 0xb00327c8U + w[26];
			b += t1;
			f = t1 + (rrot(g, 2) ^ rrot(g, 13) ^ rrot(g, 22)) + ((a & h) | (g & (a | h)));
			//
			t1 = e + (rrot(b, 6) ^ rrot(b, 11) ^ rrot(b, 25)) + (d ^ (b & (c ^ d))) + 0xbf597fc7U + w[27];
			a += t1;
			e = t1 + (rrot(f, 2) ^ rrot(f, 13) ^ rrot(f, 22)) + ((h & g) | (f & (h | g)));
			//
			t1 = d + (rrot(a, 6) ^ rrot(a, 11) ^ rrot(a, 25)) + (c ^ (a & (b ^ c))) + 0xc6e00bf3U + w[28];
			h += t1;
			d = t1 + (rrot(e, 2) ^ rrot(e, 13) ^ rrot(e, 22)) + ((g & f) | (e & (g | f)));
			//
			t1 = c + (rrot(h, 6) ^ rrot(h, 11) ^ rrot(h, 25)) + (b ^ (h & (a ^ b))) + 0xd5a79147U + w[29];
			g += t1;
			c = t1 + (rrot(d, 2) ^ rrot(d, 13) ^ rrot(d, 22)) + ((f & e) | (d & (f | e)));
			//
			t1 = b + (rrot(g, 6) ^ rrot(g, 11) ^ rrot(g, 25)) + (a ^ (g & (h ^ a))) + 0x06ca6351U + w[30];
			f += t1;
			b = t1 + (rrot(c, 2) ^ rrot(c, 13) ^ rrot(c, 22)) + ((e & d) | (c & (e | d)));
			//
			t1 = a + (rrot(f, 6) ^ rrot(f, 11) ^ rrot(f, 25)) + (h ^ (f & (g ^ h))) + 0x14292967U + w[31];
			e += t1;
			a = t1 + (rrot(b, 2) ^ rrot(b, 13) ^ rrot(b, 22)) + ((d & c) | (b & (d | c)));
			//
			t1 = h + (rrot(e, 6) ^ rrot(e, 11) ^ rrot(e, 25)) + (g ^ (e & (f ^ g))) + 0x27b70a85U + w[32];
			d += t1;
			h = t1 + (rrot(a, 2) ^ rrot(a, 13) ^ rrot(a, 22)) + ((c & b) | (a & (c | b)));
			//
			t1 = g + (rrot(d, 6) ^ rrot(d, 11) ^ rrot(d, 25)) + (f ^ (d & (e ^ f))) + 0x2e1b2138U + w[33];
			c += t1;
			g = t1 + (rrot(h, 2) ^ rrot(h, 13) ^ rrot(h, 22)) + ((b & a) | (h & (b | a)));
			//
			t1 = f + (rrot(c, 6) ^ rrot(c, 11) ^ rrot(c, 25)) + (e ^ (c & (d ^ e))) + 0x4d2c6dfcU + w[34];
			b += t1;
			f = t1 + (rrot(g, 2) ^ rrot(g, 13) ^ rrot(g, 22)) + ((a & h) | (g & (a | h)));
			//
			t1 = e + (rrot(b, 6) ^ rrot(b, 11) ^ rrot(b, 25)) + (d ^ (b & (c ^ d))) + 0x53380d13U + w[35];
			a += t1;
			e = t1 + (rrot(f, 2) ^ rrot(f, 13) ^ rrot(f, 22)) + ((h & g) | (f & (h | g)));
			//
			t1 = d + (rrot(a, 6) ^ rrot(a, 11) ^ rrot(a, 25)) + (c ^ (a & (b ^ c))) + 0x650a7354U + w[36];
			h += t1;
			d = t1 + (rrot(e, 2) ^ rrot(e, 13) ^ rrot(e, 22)) + ((g & f) | (e & (g | f)));
			//
			t1 = c + (rrot(h, 6) ^ rrot(h, 11) ^ rrot(h, 25)) + (b ^ (h & (a ^ b))) + 0x766a0abbU + w[37];
			g += t1;
			c = t1 + (rrot(d, 2) ^ rrot(d, 13) ^ rrot(d, 22)) + ((f & e) | (d & (f | e)));
			//
			t1 = b + (rrot(g, 6) ^ rrot(g, 11) ^ rrot(g, 25)) + (a ^ (g & (h ^ a))) + 0x81c2c92eU + w[38];
			f += t1;
			b = t1 + (rrot(c, 2) ^ rrot(c, 13) ^ rrot(c, 22)) + ((e & d) | (c & (e | d)));
			//
			t1 = a + (rrot(f, 6) ^ rrot(f, 11) ^ rrot(f, 25)) + (h ^ (f & (g ^ h))) + 0x92722c85U + w[39];
			e += t1;
			a = t1 + (rrot(b, 2) ^ rrot(b, 13) ^ rrot(b, 22)) + ((d & c) | (b & (d | c)));
			//
			t1 = h + (rrot(e, 6) ^ rrot(e, 11) ^ rrot(e, 25)) + (g ^ (e & (f ^ g))) + 0xa2bfe8a1U + w[40];
			d += t1;
			h = t1 + (rrot(a, 2) ^ rrot(a, 13) ^ rrot(a, 22)) + ((c & b) | (a & (c | b)));
			//
			t1 = g + (rrot(d, 6) ^ rrot(d, 11) ^ rrot(d, 25)) + (f ^ (d & (e ^ f))) + 0xa81a664bU + w[41];
			c += t1;
			g = t1 + (rrot(h, 2) ^ rrot(h, 13) ^ rrot(h, 22)) + ((b & a) | (h & (b | a)));
			//
			t1 = f + (rrot(c, 6) ^ rrot(c, 11) ^ rrot(c, 25)) + (e ^ (c & (d ^ e))) + 0xc24b8b70U + w[42];
			b += t1;
			f = t1 + (rrot(g, 2) ^ rrot(g, 13) ^ rrot(g, 22)) + ((a & h) | (g & (a | h)));
			//
			t1 = e + (rrot(b, 6) ^ rrot(b, 11) ^ rrot(b, 25)) + (d ^ (b & (c ^ d))) + 0xc76c51a3U + w[43];
			a += t1;
			e = t1 + (rrot(f, 2) ^ rrot(f, 13) ^ rrot(f, 22)) + ((h & g) | (f & (h | g)));
			//
			t1 = d + (rrot(a, 6) ^ rrot(a, 11) ^ rrot(a, 25)) + (c ^ (a & (b ^ c))) + 0xd192e819U + w[44];
			h += t1;
			d = t1 + (rrot(e, 2) ^ rrot(e, 13) ^ rrot(e, 22)) + ((g & f) | (e & (g | f)));
			//
			t1 = c + (rrot(h, 6) ^ rrot(h, 11) ^ rrot(h, 25)) + (b ^ (h & (a ^ b))) + 0xd6990624U + w[45];
			g += t1;
			c = t1 + (rrot(d, 2) ^ rrot(d, 13) ^ rrot(d, 22)) + ((f & e) | (d & (f | e)));
			//
			t1 = b + (rrot(g, 6) ^ rrot(g, 11) ^ rrot(g, 25)) + (a ^ (g & (h ^ a))) + 0xf40e3585U + w[46];
			f += t1;
			b = t1 + (rrot(c, 2) ^ rrot(c, 13) ^ rrot(c, 22)) + ((e & d) | (c & (e | d)));
			//
			t1 = a + (rrot(f, 6) ^ rrot(f, 11) ^ rrot(f, 25)) + (h ^ (f & (g ^ h))) + 0x106aa070U + w[47];
			e += t1;
			a = t1 + (rrot(b, 2) ^ rrot(b, 13) ^ rrot(b, 22)) + ((d & c) | (b & (d | c)));
			//
			t1 = h + (rrot(e, 6) ^ rrot(e, 11) ^ rrot(e, 25)) + (g ^ (e & (f ^ g))) + 0x19a4c116U + w[48];
			d += t1;
			h = t1 + (rrot(a, 2) ^ rrot(a, 13) ^ rrot(a, 22)) + ((c & b) | (a & (c | b)));
			//
			t1 = g + (rrot(d, 6) ^ rrot(d, 11) ^ rrot(d, 25)) + (f ^ (d & (e ^ f))) + 0x1e376c08U + w[49];
			c += t1;
			g = t1 + (rrot(h, 2) ^ rrot(h, 13) ^ rrot(h, 22)) + ((b & a) | (h & (b | a)));
			//
			t1 = f + (rrot(c, 6) ^ rrot(c, 11) ^ rrot(c, 25)) + (e ^ (c & (d ^ e))) + 0x2748774cU + w[50];
			b += t1;
			f = t1 + (rrot(g, 2) ^ rrot(g, 13) ^ rrot(g, 22)) + ((a & h) | (g & (a | h)));
			//
			t1 = e + (rrot(b, 6) ^ rrot(b, 11) ^ rrot(b, 25)) + (d ^ (b & (c ^ d))) + 0x34b0bcb5U + w[51];
			a += t1;
			e = t1 + (rrot(f, 2) ^ rrot(f, 13) ^ rrot(f, 22)) + ((h & g) | (f & (h | g)));
			//
			t1 = d + (rrot(a, 6) ^ rrot(a, 11) ^ rrot(a, 25)) + (c ^ (a & (b ^ c))) + 0x391c0cb3U + w[52];
			h += t1;
			d = t1 + (rrot(e, 2) ^ rrot(e, 13) ^ rrot(e, 22)) + ((g & f) | (e & (g | f)));
			//
			t1 = c + (rrot(h, 6) ^ rrot(h, 11) ^ rrot(h, 25)) + (b ^ (h & (a ^ b))) + 0x4ed8aa4aU + w[53];
			g += t1;
			c = t1 + (rrot(d, 2) ^ rrot(d, 13) ^ rrot(d, 22)) + ((f & e) | (d & (f | e)));
			//
			t1 = b + (rrot(g, 6) ^ rrot(g, 11) ^ rrot(g, 25)) + (a ^ (g & (h ^ a))) + 0x5b9cca4fU + w[54];
			f += t1;
			b = t1 + (rrot(c, 2) ^ rrot(c, 13) ^ rrot(c, 22)) + ((e & d) | (c & (e | d)));
			//
			t1 = a + (rrot(f, 6) ^ rrot(f, 11) ^ rrot(f, 25)) + (h ^ (f & (g ^ h))) + 0x682e6ff3U + w[55];
			e += t1;
			a = t1 + (rrot(b, 2) ^ rrot(b, 13) ^ rrot(b, 22)) + ((d & c) | (b & (d | c)));
			//
			t1 = h + (rrot(e, 6) ^ rrot(e, 11) ^ rrot(e, 25)) + (g ^ (e & (f ^ g))) + 0x748f82eeU + w[56];
			d += t1;
			h = t1 + (rrot(a, 2) ^ rrot(a, 13) ^ rrot(a, 22)) + ((c & b) | (a & (c | b)));
			//
			c += g + (rrot(d, 6) ^ rrot(d, 11) ^ rrot(d, 25)) + (f ^ (d & (e ^ f))) + 0x78a5636fU + w[57];
			//
			b += f + (rrot(c, 6) ^ rrot(c, 11) ^ rrot(c, 25)) + (e ^ (c & (d ^ e))) + 0x84c87814U + w[58];
			//
			a += e + (rrot(b, 6) ^ rrot(b, 11) ^ rrot(b, 25)) + (d ^ (b & (c ^ d))) + 0x8cc70208U + w[43] + w[52] + (rrot(w[44], 7) ^ rrot(w[44], 18) ^ (w[44] >> 3)) + (rrot(w[57], 17) ^ rrot(w[57], 19) ^ (w[57] >> 10));
			//
			h += d + (rrot(a, 6) ^ rrot(a, 11) ^ rrot(a, 25)) + (c ^ (a & (b ^ c))) + 0x90befffaU + w[44] + w[53] + (rrot(w[45], 7) ^ rrot(w[45], 18) ^ (w[45] >> 3)) + (rrot(w[58], 17) ^ rrot(w[58], 19) ^ (w[58] >> 10));
			//
			if (h == 0xa41f32e7)
			{
				uint32_t tmp = atomicCAS(result, 0xffffffff, nonce);
				if (tmp != 0xffffffff)
					result[1] = nonce;
			}
		} // nonce loop
	} // if thread<threads
}

__host__
void bitcoin_midstate(const uint32_t *data, uint32_t *midstate)
{
	int i;
	uint32_t s0, s1, t1, t2, maj, ch, a, b, c, d, e, f, g, h;
	uint32_t w[64];

	const uint32_t k[64] = {
		0x428a2f98U, 0x71374491U, 0xb5c0fbcfU, 0xe9b5dba5U, 0x3956c25bU, 0x59f111f1U, 0x923f82a4U, 0xab1c5ed5U,
		0xd807aa98U, 0x12835b01U, 0x243185beU, 0x550c7dc3U, 0x72be5d74U, 0x80deb1feU, 0x9bdc06a7U, 0xc19bf174U,
		0xe49b69c1U, 0xefbe4786U, 0x0fc19dc6U, 0x240ca1ccU, 0x2de92c6fU, 0x4a7484aaU, 0x5cb0a9dcU, 0x76f988daU,
		0x983e5152U, 0xa831c66dU, 0xb00327c8U, 0xbf597fc7U, 0xc6e00bf3U, 0xd5a79147U, 0x06ca6351U, 0x14292967U,
		0x27b70a85U, 0x2e1b2138U, 0x4d2c6dfcU, 0x53380d13U, 0x650a7354U, 0x766a0abbU, 0x81c2c92eU, 0x92722c85U,
		0xa2bfe8a1U, 0xa81a664bU, 0xc24b8b70U, 0xc76c51a3U, 0xd192e819U, 0xd6990624U, 0xf40e3585U, 0x106aa070U,
		0x19a4c116U, 0x1e376c08U, 0x2748774cU, 0x34b0bcb5U, 0x391c0cb3U, 0x4ed8aa4aU, 0x5b9cca4fU, 0x682e6ff3U,
		0x748f82eeU, 0x78a5636fU, 0x84c87814U, 0x8cc70208U, 0x90befffaU, 0xa4506cebU, 0xbef9a3f7U, 0xc67178f2U
	};
	const uint32_t hc[8] = {
		0x6a09e667U, 0xbb67ae85U, 0x3c6ef372U, 0xa54ff53aU,
		0x510e527fU, 0x9b05688cU, 0x1f83d9abU, 0x5be0cd19U
	};

	for (i = 0; i <= 15; i++)
	{
		w[i] = data[i];
	}
	for (i = 16; i <= 63; i++)
	{
		s0 = rrot(w[i - 15], 7) ^ rrot(w[i - 15], 18) ^ (w[i - 15] >> 3);
		s1 = rrot(w[i - 2], 17) ^ rrot(w[i - 2], 19) ^ (w[i - 2] >> 10);
		w[i] = w[i - 16] + s0 + w[i - 7] + s1;
	}
	a = hc[0];
	b = hc[1];
	c = hc[2];
	d = hc[3];
	e = hc[4];
	f = hc[5];
	g = hc[6];
	h = hc[7];
	for (i = 0; i <= 63; i++)
	{
		s0 = rrot(a, 2) ^ rrot(a, 13) ^ rrot(a, 22);
		maj = (a & b) ^ (a & c) ^ (b & c);
		t2 = s0 + maj;
		s1 = rrot(e, 6) ^ rrot(e, 11) ^ rrot(e, 25);
		ch = (e & f) ^ ((~e) & g);
		t1 = h + s1 + ch + k[i] + w[i];
		h = g;
		g = f;
		f = e;
		e = d + t1;
		d = c;
		c = b;
		b = a;
		a = t1 + t2;
	}
	midstate[0] = a + hc[0];
	midstate[1] = b + hc[1];
	midstate[2] = c + hc[2];
	midstate[3] = d + hc[3];
	midstate[4] = e + hc[4];
	midstate[5] = f + hc[5];
	midstate[6] = g + hc[6];
	midstate[7] = h + hc[7];
}

__host__
void bitcoin_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce, const uint32_t *const ms, uint32_t merkle, uint32_t time, uint32_t compacttarget, uint32_t *const h_nounce)
{
	uint32_t b2, c2, d2, f2, g2, h2, t1, w16, w17, t1c, t2c, w16rot, w17rot;

	hipMemset(d_result[thr_id], 0xff, 2 * sizeof(uint32_t));

	t1 = ms[7] + (rrot(ms[4], 6) ^ rrot(ms[4], 11) ^ rrot(ms[4], 25)) + (ms[6] ^ (ms[4] & (ms[5] ^ ms[6]))) + 0x428a2f98U + merkle;
	d2 = ms[3] + t1;
	h2 = t1 + (rrot(ms[0], 2) ^ rrot(ms[0], 13) ^ rrot(ms[0], 22)) + ((ms[2] & ms[1]) | (ms[0] & (ms[2] | ms[1])));
	//
	t1 = ms[6] + (rrot(d2, 6) ^ rrot(d2, 11) ^ rrot(d2, 25)) + (ms[5] ^ (d2 & (ms[4] ^ ms[5]))) + 0x71374491U + time;
	c2 = ms[2] + t1;
	g2 = t1 + (rrot(h2, 2) ^ rrot(h2, 13) ^ rrot(h2, 22)) + ((ms[1] & ms[0]) | (h2 & (ms[1] | ms[0])));
	//
	t1 = ms[5] + (rrot(c2, 6) ^ rrot(c2, 11) ^ rrot(c2, 25)) + (ms[4] ^ (c2 & (d2 ^ ms[4]))) + 0xb5c0fbcfU + compacttarget;
	b2 = ms[1] + t1;
	f2 = t1 + (rrot(g2, 2) ^ rrot(g2, 13) ^ rrot(g2, 22)) + ((ms[0] & h2) | (g2 & (ms[0] | h2)));

	w16 = merkle + (rrot(time, 7) ^ rrot(time, 18) ^ (time >> 3));
	w16rot = (rrot(w16, 17) ^ rrot(w16, 19) ^ (w16 >> 10)) + compacttarget;
	w17 = time + (rrot(compacttarget, 7) ^ rrot(compacttarget, 18) ^ (compacttarget >> 3)) + 0x01100000U;
	w17rot = (rrot(w17, 17) ^ rrot(w17, 19) ^ (w17 >> 10)) + 0x11002000U;
	t2c = (rrot(f2, 2) ^ rrot(f2, 13) ^ rrot(f2, 22)) + ((h2 & g2) | (f2 & (h2 | g2)));
	t1c = ms[4] + (rrot(b2, 6) ^ rrot(b2, 11) ^ rrot(b2, 25)) + (d2 ^ (b2 & (c2 ^ d2))) + 0xe9b5dba5U;

	dim3 grid((threads + TPB*NONCES_PER_THREAD - 1) / TPB / NONCES_PER_THREAD);
	dim3 block(TPB);
	bitcoin_gpu_hash << <grid, block >> > (threads, startNounce, d_result[thr_id], t1c, t2c, w16, w16rot, w17, w17rot, b2, c2, d2, f2, g2, h2, ms[0], ms[1], ms[2], ms[3], ms[4], ms[5], ms[6], ms[7], compacttarget);
	CUDA_SAFE_CALL(hipMemcpy(h_nounce, d_result[thr_id], 2 * sizeof(uint32_t), hipMemcpyDeviceToHost));
}

__host__
void bitcoin_cpu_init(int thr_id)
{
	CUDA_SAFE_CALL(hipMalloc(&d_result[thr_id], 4 * sizeof(uint32_t)));
}
