#include "hip/hip_runtime.h"
/*
 * luffa_for_32.c
 * Version 2.0 (Sep 15th 2009)
 *
 * Copyright (C) 2008-2009 Hitachi, Ltd. All rights reserved.
 *
 * Hitachi, Ltd. is the owner of this software and hereby grant
 * the U.S. Government and any interested party the right to use
 * this software for the purposes of the SHA-3 evaluation process,
 * notwithstanding that this software is copyrighted.
 *
 * THE SOFTWARE IS PROVIDED "AS IS" AND THE AUTHOR DISCLAIMS ALL WARRANTIES
 * WITH REGARD TO THIS SOFTWARE INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY AND FITNESS. IN NO EVENT SHALL THE AUTHOR BE LIABLE FOR
 * ANY SPECIAL, DIRECT, INDIRECT, OR CONSEQUENTIAL DAMAGES OR ANY DAMAGES
 * WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS, WHETHER IN AN
 * ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS ACTION, ARISING OUT OF
 * OR IN CONNECTION WITH THE USE OR PERFORMANCE OF THIS SOFTWARE.
 */

#include "cuda_helper.h"

typedef unsigned char BitSequence;

typedef struct {
    uint32_t buffer[8]; /* Buffer to be hashed */
    uint32_t chainv[40];   /* Chaining values */
} hashState;

#define MULT2(a,j)\
    tmp = a[7+(8*j)];\
    a[7+(8*j)] = a[6+(8*j)];\
    a[6+(8*j)] = a[5+(8*j)];\
    a[5+(8*j)] = a[4+(8*j)];\
    a[4+(8*j)] = a[3+(8*j)] ^ tmp;\
    a[3+(8*j)] = a[2+(8*j)] ^ tmp;\
    a[2+(8*j)] = a[1+(8*j)];\
    a[1+(8*j)] = a[0+(8*j)] ^ tmp;\
    a[0+(8*j)] = tmp;

#if __CUDA_ARCH__ < 350
#define LROT(x,bits) ((x << bits) | (x >> (32 - bits)))
#else
#define LROT(x, bits) __funnelshift_l(x, x, bits)
#endif

#define TWEAK(a0,a1,a2,a3,j)\
    a0 = LROT(a0,j);\
    a1 = LROT(a1,j);\
    a2 = LROT(a2,j);\
    a3 = LROT(a3,j);

#define STEP(c0,c1)\
    SUBCRUMB(chainv[0],chainv[1],chainv[2],chainv[3],tmp);\
    SUBCRUMB(chainv[5],chainv[6],chainv[7],chainv[4],tmp);\
    MIXWORD(chainv[0],chainv[4]);\
    MIXWORD(chainv[1],chainv[5]);\
    MIXWORD(chainv[2],chainv[6]);\
    MIXWORD(chainv[3],chainv[7]);\
    ADD_CONSTANT(chainv[0],chainv[4],c0,c1);

#define SUBCRUMB(a0,a1,a2,a3,a4)\
    a4  = a0;\
    a0 |= a1;\
    a2 ^= a3;\
    a1  = ~a1;\
    a0 ^= a3;\
    a3 &= a4;\
    a1 ^= a3;\
    a3 ^= a2;\
    a2 &= a0;\
    a0  = ~a0;\
    a2 ^= a1;\
    a1 |= a3;\
    a4 ^= a1;\
    a3 ^= a2;\
    a2 &= a1;\
    a1 ^= a0;\
    a0  = a4;

#define MIXWORD(a0,a4)\
    a4 ^= a0;\
    a0  = LROT(a0,2);\
    a0 ^= a4;\
    a4  = LROT(a4,14);\
    a4 ^= a0;\
    a0  = LROT(a0,10);\
    a0 ^= a4;\
    a4  = LROT(a4,1);

#define ADD_CONSTANT(a0,b0,c0,c1)\
    a0 ^= c0;\
    b0 ^= c1;

// Precalculated chaining values
__device__ __constant__ uint32_t c_IV[40] =
{ 0x8bb0a761, 0xc2e4aa8b, 0x2d539bc9, 0x381408f8,
0x478f6633, 0x255a46ff, 0x581c37f7, 0x601c2e8e,
0x266c5f9d, 0xc34715d8, 0x8900670e, 0x51a540be,
0xe4ce69fb, 0x5089f4d4, 0x3cc0a506, 0x609bcb02,
0xa4e3cd82, 0xd24fd6ca, 0xc0f196dc, 0xcf41eafe,
0x0ff2e673, 0x303804f2, 0xa7b3cd48, 0x677addd4,
0x66e66a8a, 0x2303208f, 0x486dafb4, 0xc0d37dc6,
0x634d15af, 0xe5af6747, 0x10af7e38, 0xee7e6428,
0x01262e5d, 0xc92c2e64, 0x82fee966, 0xcea738d3,
0x867de2b0, 0xe0714818, 0xda6e831f, 0xa7062529};



/* old chaining values
__device__ __constant__ uint32_t c_IV[40] = {
    0x6d251e69,0x44b051e0,0x4eaa6fb4,0xdbf78465,
    0x6e292011,0x90152df4,0xee058139,0xdef610bb,
    0xc3b44b95,0xd9d2f256,0x70eee9a0,0xde099fa3,
    0x5d9b0557,0x8fc944b3,0xcf1ccf0e,0x746cd581,
    0xf7efc89d,0x5dba5781,0x04016ce5,0xad659c05,
    0x0306194f,0x666d1836,0x24aa230a,0x8b264ae7,
    0x858075d5,0x36d79cce,0xe571f7d7,0x204b1f67,
    0x35870c6a,0x57e9e923,0x14bcb808,0x7cde72ce,
    0x6c68e9be,0x5ec41e22,0xc825b7c7,0xaffb4363,
    0xf5df3999,0x0fc688f1,0xb07224cc,0x03e86cea};
*/


__device__ __constant__ uint32_t c_CNS[80] = {
    0x303994a6,0xe0337818,0xc0e65299,0x441ba90d,
    0x6cc33a12,0x7f34d442,0xdc56983e,0x9389217f,
    0x1e00108f,0xe5a8bce6,0x7800423d,0x5274baf4,
    0x8f5b7882,0x26889ba7,0x96e1db12,0x9a226e9d,
    0xb6de10ed,0x01685f3d,0x70f47aae,0x05a17cf4,
    0x0707a3d4,0xbd09caca,0x1c1e8f51,0xf4272b28,
    0x707a3d45,0x144ae5cc,0xaeb28562,0xfaa7ae2b,
    0xbaca1589,0x2e48f1c1,0x40a46f3e,0xb923c704,
    0xfc20d9d2,0xe25e72c1,0x34552e25,0xe623bb72,
    0x7ad8818f,0x5c58a4a4,0x8438764a,0x1e38e2e7,
    0xbb6de032,0x78e38b9d,0xedb780c8,0x27586719,
    0xd9847356,0x36eda57f,0xa2c78434,0x703aace7,
    0xb213afa5,0xe028c9bf,0xc84ebe95,0x44756f91,
    0x4e608a22,0x7e8fce32,0x56d858fe,0x956548be,
    0x343b138f,0xfe191be2,0xd0ec4e3d,0x3cb226e5,
    0x2ceb4882,0x5944a28e,0xb3ad2208,0xa1c4c355,
    0xf0d2e9e3,0x5090d577,0xac11d7fa,0x2d1925ab,
    0x1bcb66f2,0xb46496ac,0x6f2d9bc9,0xd1925ab0,
    0x78602649,0x29131ab6,0x8edae952,0x0fc053c3,
    0x3b6ba548,0x3f014f0c,0xedae9520,0xfc053c31};


/***************************************************/
__device__ __forceinline__
void rnd512(hashState *state)
{
    int i,j;
    uint32_t t[40];
    uint32_t chainv[8];
    uint32_t tmp;

#pragma unroll 8
    for(i=0;i<8;i++) 
	{
		t[i] = 0;
#pragma unroll 5
        for(j=0;j<5;j++) 
		{
           t[i] ^= state->chainv[i+8*j];
        }
	}

    MULT2(t, 0);

#pragma unroll 5
    for(j=0;j<5;j++) {
#pragma unroll 8
        for(i=0;i<8;i++) {
            state->chainv[i+8*j] ^= t[i];
        }
    }

#pragma unroll 5
    for(j=0;j<5;j++) {
#pragma unroll 8
        for(i=0;i<8;i++) {
            t[i+8*j] = state->chainv[i+8*j];
        }
    }

#pragma unroll 5
    for(j=0;j<5;j++) {
        MULT2(state->chainv, j);
    }

#pragma unroll 5
    for(j=0;j<5;j++) {
#pragma unroll 8
        for(i=0;i<8;i++) {
            state->chainv[8*j+i] ^= t[8*((j+1)%5)+i];
        }
    }

#pragma unroll 5
    for(j=0;j<5;j++) {
#pragma unroll 8
        for(i=0;i<8;i++) {
            t[i+8*j] = state->chainv[i+8*j];
        }
    }

#pragma unroll 5
    for(j=0;j<5;j++) {
        MULT2(state->chainv, j);
    }

#pragma unroll 5
    for(j=0;j<5;j++) {
#pragma unroll 8
        for(i=0;i<8;i++) {
            state->chainv[8*j+i] ^= t[8*((j+4)%5)+i];
        }
    }

#pragma unroll 5
    for(j=0;j<5;j++) {
#pragma unroll 8
        for(i=0;i<8;i++) {
            state->chainv[i+8*j] ^= state->buffer[i];
        }
        MULT2(state->buffer, 0);
    }

#pragma unroll 8
    for(i=0;i<8;i++) {
        chainv[i] = state->chainv[i];
    }

#pragma unroll 8
    for(i=0;i<8;i++) {
        STEP(c_CNS[(2*i)],c_CNS[(2*i)+1]);
    }

#pragma unroll 8
    for(i=0;i<8;i++) {
        state->chainv[i] = chainv[i];
        chainv[i] = state->chainv[i+8];
    }

    TWEAK(chainv[4],chainv[5],chainv[6],chainv[7],1);

#pragma unroll 8
    for(i=0;i<8;i++) {
        STEP(c_CNS[(2*i)+16],c_CNS[(2*i)+16+1]);
    }

#pragma unroll 8
    for(i=0;i<8;i++) {
        state->chainv[i+8] = chainv[i];
        chainv[i] = state->chainv[i+16];
    }

    TWEAK(chainv[4],chainv[5],chainv[6],chainv[7],2);

#pragma unroll 8
    for(i=0;i<8;i++) {
        STEP(c_CNS[(2*i)+32],c_CNS[(2*i)+32+1]);
    }

#pragma unroll 8
    for(i=0;i<8;i++) {
        state->chainv[i+16] = chainv[i];
        chainv[i] = state->chainv[i+24];
    }

    TWEAK(chainv[4],chainv[5],chainv[6],chainv[7],3);

#pragma unroll 8
    for(i=0;i<8;i++) {
        STEP(c_CNS[(2*i)+48],c_CNS[(2*i)+48+1]);
    }

#pragma unroll 8
    for(i=0;i<8;i++) {
        state->chainv[i+24] = chainv[i];
        chainv[i] = state->chainv[i+32];
    }

    TWEAK(chainv[4],chainv[5],chainv[6],chainv[7],4);

#pragma unroll 8
    for(i=0;i<8;i++) {
        STEP(c_CNS[(2*i)+64],c_CNS[(2*i)+64+1]);
    }

#pragma unroll 8
    for(i=0;i<8;i++) {
        state->chainv[i+32] = chainv[i];
    }
}
__device__ __forceinline__
void rnd512_first(uint32_t state[40], uint32_t buffer[8])
{
	int i, j;
	uint32_t chainv[8];
	uint32_t tmp;

#pragma unroll 5
	for (j = 0; j<5; j++) {
		state[0 + 8 * j] ^= buffer[0];

#pragma unroll 7
		for (i = 1; i<8; i++) {
			state[i + 8 * j] ^= buffer[i];
		}
		MULT2(buffer, 0);
	}

#pragma unroll 8
	for (i = 0; i<8; i++) {
		chainv[i] = state[i];
	}

#pragma unroll 8
	for (i = 0; i<8; i++) {
		STEP(c_CNS[(2 * i)], c_CNS[(2 * i) + 1]);
	}

#pragma unroll 8
	for (i = 0; i<8; i++) {
		state[i] = chainv[i];
		chainv[i] = state[i + 8];
	}

	TWEAK(chainv[4], chainv[5], chainv[6], chainv[7], 1);

#pragma unroll 8
	for (i = 0; i<8; i++) {
		STEP(c_CNS[(2 * i) + 16], c_CNS[(2 * i) + 16 + 1]);
	}

#pragma unroll 8
	for (i = 0; i<8; i++) {
		state[i + 8] = chainv[i];
		chainv[i] = state[i + 16];
	}

	TWEAK(chainv[4], chainv[5], chainv[6], chainv[7], 2);

#pragma unroll 8
	for (i = 0; i<8; i++) {
		STEP(c_CNS[(2 * i) + 32], c_CNS[(2 * i) + 32 + 1]);
	}

#pragma unroll 8
	for (i = 0; i<8; i++) {
		state[i + 16] = chainv[i];
		chainv[i] = state[i + 24];
	}

	TWEAK(chainv[4], chainv[5], chainv[6], chainv[7], 3);

#pragma unroll 8
	for (i = 0; i<8; i++) {
		STEP(c_CNS[(2 * i) + 48], c_CNS[(2 * i) + 48 + 1]);
	}

#pragma unroll 8
	for (i = 0; i<8; i++) {
		state[i + 24] = chainv[i];
		chainv[i] = state[i + 32];
	}

	TWEAK(chainv[4], chainv[5], chainv[6], chainv[7], 4);

#pragma unroll 8
	for (i = 0; i<8; i++) {
		STEP(c_CNS[(2 * i) + 64], c_CNS[(2 * i) + 64 + 1]);
	}

#pragma unroll 8
	for (i = 0; i<8; i++) {
		state[i + 32] = chainv[i];
	}
}

/***************************************************/
__device__ __forceinline__
void rnd512_nullhash(uint32_t *state)
{
	int i, j;
	uint32_t t[40];
	uint32_t chainv[8];
	uint32_t tmp;

#pragma unroll 8
	for (i = 0; i<8; i++) {
		t[i] = state[i + 8 * 0];
#pragma unroll 4
		for (j = 1; j<5; j++) {
			t[i] ^= state[i + 8 * j];
		}
	}

	MULT2(t, 0);

#pragma unroll 5
	for (j = 0; j<5; j++) {
#pragma unroll 8
		for (i = 0; i<8; i++) {
			state[i + 8 * j] ^= t[i];
		}
	}

#pragma unroll 5
	for (j = 0; j<5; j++) {
#pragma unroll 8
		for (i = 0; i<8; i++) {
			t[i + 8 * j] = state[i + 8 * j];
		}
	}

#pragma unroll 5
	for (j = 0; j<5; j++) {
		MULT2(state, j);
	}

#pragma unroll 5
	for (j = 0; j<5; j++) {
#pragma unroll 8
		for (i = 0; i<8; i++) {
			state[8 * j + i] ^= t[8 * ((j + 1) % 5) + i];
		}
	}

#pragma unroll 5
	for (j = 0; j<5; j++) {
#pragma unroll 8
		for (i = 0; i<8; i++) {
			t[i + 8 * j] = state[i + 8 * j];
		}
	}

#pragma unroll 5
	for (j = 0; j<5; j++) {
		MULT2(state, j);
	}

#pragma unroll 5
	for (j = 0; j<5; j++) {
#pragma unroll 8
		for (i = 0; i<8; i++) {
			state[8 * j + i] ^= t[8 * ((j + 4) % 5) + i];
		}
	}

#pragma unroll 8
	for (i = 0; i<8; i++) {
		chainv[i] = state[i];
	}

#pragma unroll 8
	for (i = 0; i<8; i++) {
		STEP(c_CNS[(2 * i)], c_CNS[(2 * i) + 1]);
	}

#pragma unroll 8
	for (i = 0; i<8; i++) {
		state[i] = chainv[i];
		chainv[i] = state[i + 8];
	}

	TWEAK(chainv[4], chainv[5], chainv[6], chainv[7], 1);

#pragma unroll 8
	for (i = 0; i<8; i++) {
		STEP(c_CNS[(2 * i) + 16], c_CNS[(2 * i) + 16 + 1]);
	}

#pragma unroll 8
	for (i = 0; i<8; i++) {
		state[i + 8] = chainv[i];
		chainv[i] = state[i + 16];
	}

	TWEAK(chainv[4], chainv[5], chainv[6], chainv[7], 2);

#pragma unroll 8
	for (i = 0; i<8; i++) {
		STEP(c_CNS[(2 * i) + 32], c_CNS[(2 * i) + 32 + 1]);
	}

#pragma unroll 8
	for (i = 0; i<8; i++) {
		state[i + 16] = chainv[i];
		chainv[i] = state[i + 24];
	}

	TWEAK(chainv[4], chainv[5], chainv[6], chainv[7], 3);

#pragma unroll 8
	for (i = 0; i<8; i++) {
		STEP(c_CNS[(2 * i) + 48], c_CNS[(2 * i) + 48 + 1]);
	}

#pragma unroll 8
	for (i = 0; i<8; i++) {
		state[i + 24] = chainv[i];
		chainv[i] = state[i + 32];
	}

	TWEAK(chainv[4], chainv[5], chainv[6], chainv[7], 4);

#pragma unroll 8
	for (i = 0; i<8; i++) {
		STEP(c_CNS[(2 * i) + 64], c_CNS[(2 * i) + 64 + 1]);
	}

#pragma unroll 8
	for (i = 0; i<8; i++) {
		state[i + 32] = chainv[i];
	}
}
__device__ __forceinline__
void Update512(hashState *state, const uint32_t*data)
{
#pragma unroll 8
	for (int i = 0; i < 8; i++) state->buffer[i] = cuda_swab32(data[i]);
    rnd512_first(state->chainv, state->buffer);

#pragma unroll 8
	for (int i = 0; i < 8; i++) state->buffer[i] = cuda_swab32(data[i + 8]);
    rnd512(state);
}


/***************************************************/
__device__ __forceinline__
void finalization512(hashState *state, uint32_t *b)
{
    int i,j;

    state->buffer[0] = 0x80000000;
	#pragma unroll 7
    for(int i=1;i<8;i++) state->buffer[i] = 0;
	rnd512(state);

    /*---- blank round with m=0 ----*/
	rnd512_nullhash(state->chainv);

#pragma unroll 8
    for(i=0;i<8;i++) {
		b[i] = state->chainv[i + 8 * 0];
#pragma unroll 4
        for(j=1;j<5;j++) {
            b[i] ^= state->chainv[i+8*j];
        }
        b[i] = cuda_swab32((b[i]));
    }

	rnd512_nullhash(state->chainv);

#pragma unroll 8
    for(i=0;i<8;i++) {
		b[8 + i] = state->chainv[i + 8 * 0];
#pragma unroll 4
        for(j=1;j<5;j++) {
            b[8+i] ^= state->chainv[i+8*j];
        }
        b[8 + i] = cuda_swab32((b[8 + i]));
    }
}


typedef unsigned char BitSequence;

#define CUBEHASH_ROUNDS 16 /* this is r for CubeHashr/b */
#define CUBEHASH_BLOCKBYTES 32 /* this is b for CubeHashr/b */

#if __CUDA_ARCH__ < 350
#define LROT(x,bits) ((x << bits) | (x >> (32 - bits)))
#else
#define LROT(x, bits) __funnelshift_l(x, x, bits)
#endif

#define ROTATEUPWARDS7(a)  LROT(a,7)
#define ROTATEUPWARDS11(a) LROT(a,11)

#define SWAP(a,b) { uint32_t u = a; a = b; b = u; }

__device__ __constant__
static const uint32_t c_IV_512[32] = {

	0x2AEA2A61, 0x50F494D4, 0x2D538B8B,
	0x4167D83E, 0x3FEE2313, 0xC701CF8C,
	0xCC39968E, 0x50AC5695, 0x4D42C787,
	0xA647A8B3, 0x97CF0BEF, 0x825B4537,
	0xEEF864D2, 0xF22090C4, 0xD0E5CD33,
	0xA23911AE, 0xFCD398D9, 0x148FE485,
	0x1B017BEF, 0xB6444532, 0x6A536159,
	0x2FF5781C, 0x91FA7934, 0x0DBADEA9,
	0xD65C8A2B, 0xA5A70E75, 0xB1C62456,
	0xBC796576, 0x1921C8F7, 0xE7989AF1,
	0x7795D246, 0xD43E3B44
};

__device__ __forceinline__ void rrounds(uint32_t x[2][2][2][2][2])
{
	int r;
	int j;
	int k;
	int l;
	int m;

//	#pragma unroll 
	for (r = 0; r < CUBEHASH_ROUNDS; ++r) {

		/* "add x_0jklm into x_1jklmn modulo 2^32" */
#pragma unroll 2
		for (j = 0; j < 2; ++j)
#pragma unroll 2
			for (k = 0; k < 2; ++k)
#pragma unroll 2
				for (l = 0; l < 2; ++l)
#pragma unroll 2
					for (m = 0; m < 2; ++m)
						x[1][j][k][l][m] += x[0][j][k][l][m];

		/* "rotate x_0jklm upwards by 7 bits" */
#pragma unroll 2
		for (j = 0; j < 2; ++j)
#pragma unroll 2
			for (k = 0; k < 2; ++k)
#pragma unroll 2
				for (l = 0; l < 2; ++l)
#pragma unroll 2
					for (m = 0; m < 2; ++m)
						x[0][j][k][l][m] = ROTATEUPWARDS7(x[0][j][k][l][m]);

		/* "swap x_00klm with x_01klm" */
#pragma unroll 2
		for (k = 0; k < 2; ++k)
#pragma unroll 2
			for (l = 0; l < 2; ++l)
#pragma unroll 2
				for (m = 0; m < 2; ++m)
					SWAP(x[0][0][k][l][m], x[0][1][k][l][m])

					/* "xor x_1jklm into x_0jklm" */
#pragma unroll 2
					for (j = 0; j < 2; ++j)
#pragma unroll 2
						for (k = 0; k < 2; ++k)
#pragma unroll 2
							for (l = 0; l < 2; ++l)
#pragma unroll 2
								for (m = 0; m < 2; ++m)
									x[0][j][k][l][m] ^= x[1][j][k][l][m];

		/* "swap x_1jk0m with x_1jk1m" */
#pragma unroll 2
		for (j = 0; j < 2; ++j)
#pragma unroll 2
			for (k = 0; k < 2; ++k)
#pragma unroll 2
				for (m = 0; m < 2; ++m)
					SWAP(x[1][j][k][0][m], x[1][j][k][1][m])

					/* "add x_0jklm into x_1jklm modulo 2^32" */
#pragma unroll 2
					for (j = 0; j < 2; ++j)
#pragma unroll 2
						for (k = 0; k < 2; ++k)
#pragma unroll 2
							for (l = 0; l < 2; ++l)
#pragma unroll 2
								for (m = 0; m < 2; ++m)
									x[1][j][k][l][m] += x[0][j][k][l][m];

		/* "rotate x_0jklm upwards by 11 bits" */
#pragma unroll 2
		for (j = 0; j < 2; ++j)
#pragma unroll 2
			for (k = 0; k < 2; ++k)
#pragma unroll 2
				for (l = 0; l < 2; ++l)
#pragma unroll 2
					for (m = 0; m < 2; ++m)
						x[0][j][k][l][m] = ROTATEUPWARDS11(x[0][j][k][l][m]);

		/* "swap x_0j0lm with x_0j1lm" */
#pragma unroll 2
		for (j = 0; j < 2; ++j)
#pragma unroll 2
			for (l = 0; l < 2; ++l)
#pragma unroll 2
				for (m = 0; m < 2; ++m)
					SWAP(x[0][j][0][l][m], x[0][j][1][l][m])

					/* "xor x_1jklm into x_0jklm" */
#pragma unroll 2
					for (j = 0; j < 2; ++j)
#pragma unroll 2
						for (k = 0; k < 2; ++k)
#pragma unroll 2
							for (l = 0; l < 2; ++l)
#pragma unroll 2
								for (m = 0; m < 2; ++m)
									x[0][j][k][l][m] ^= x[1][j][k][l][m];

		/* "swap x_1jkl0 with x_1jkl1" */
#pragma unroll 2
		for (j = 0; j < 2; ++j)
#pragma unroll 2
			for (k = 0; k < 2; ++k)
#pragma unroll 2
				for (l = 0; l < 2; ++l)
					SWAP(x[1][j][k][l][0], x[1][j][k][l][1])

	}
}


__device__ __forceinline__ void block_tox(uint32_t *in, uint32_t x[2][2][2][2][2])
{
	int k;
	int l;
	int m;
//	uint32_t *in = block;

#pragma unroll 2
	for (k = 0; k < 2; ++k)
#pragma unroll 2
		for (l = 0; l < 2; ++l)
#pragma unroll 2
			for (m = 0; m < 2; ++m)
				x[0][0][k][l][m] ^= *in++;
}

__device__ __forceinline__ void hash_fromx(uint32_t *out, uint32_t x[2][2][2][2][2])
{
	int j;
	int k;
	int l;
	int m;
//	uint32_t *out = hash;

#pragma unroll 2
	for (j = 0; j < 2; ++j)
#pragma unroll 2
		for (k = 0; k < 2; ++k)
#pragma unroll 2
			for (l = 0; l < 2; ++l)
#pragma unroll 2
				for (m = 0; m < 2; ++m)
					*out++ = x[0][j][k][l][m];
}

void __device__ __forceinline__ Init(uint32_t x[2][2][2][2][2])
{
	int i, j, k, l, m;
#if 0
	/* "the first three state words x_00000, x_00001, x_00010" */
	/* "are set to the integers h/8, b, r respectively." */
	/* "the remaining state words are set to 0." */
#pragma unroll 2
	for (i = 0; i < 2; ++i)
#pragma unroll 2
		for (j = 0; j < 2; ++j)
#pragma unroll 2
			for (k = 0; k < 2; ++k)
#pragma unroll 2
				for (l = 0; l < 2; ++l)
#pragma unroll 2
					for (m = 0; m < 2; ++m)
						x[i][j][k][l][m] = 0;
	x[0][0][0][0][0] = 512 / 8;
	x[0][0][0][0][1] = CUBEHASH_BLOCKBYTES;
	x[0][0][0][1][0] = CUBEHASH_ROUNDS;

	/* "the state is then transformed invertibly through 10r identical rounds */
	for (i = 0; i < 10; ++i) rrounds(x);
#else
	const uint32_t *iv = c_IV_512;

#pragma unroll 2
	for (i = 0; i < 2; ++i)
#pragma unroll 2
		for (j = 0; j < 2; ++j)
#pragma unroll 2
			for (k = 0; k < 2; ++k)
#pragma unroll 2
				for (l = 0; l < 2; ++l)
#pragma unroll 2
					for (m = 0; m < 2; ++m)
						x[i][j][k][l][m] = *iv++;
#endif
}

void __device__ __forceinline__ Update32(uint32_t x[2][2][2][2][2], const uint32_t *data)
{
	/* "xor the block into the first b bytes of the state" */
	/* "and then transform the state invertibly through r identical rounds" */
	block_tox((uint32_t*)data, x);
	rrounds(x);
}

void __device__ __forceinline__ Final(uint32_t x[2][2][2][2][2], uint32_t *hashval)
{
	int i;

	/* "the integer 1 is xored into the last state word x_11111" */
	x[1][1][1][1][1] ^= 1;

	/* "the state is then transformed invertibly through 10r identical rounds" */
//	#pragma unroll 10
	for (i = 0; i < 10; ++i) rrounds(x);

	/* "output the first h/8 bytes of the state" */
	hash_fromx(hashval, x);
}


/***************************************************/
// Die Hash-Funktion
__global__
void x11_luffaCubehash512_gpu_hash_64(uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{
    uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

        int hashPosition = nounce - startNounce;
        uint32_t *Hash = (uint32_t*)&g_hash[8 * hashPosition];

        hashState state;
#pragma unroll 40
        for(int i=0;i<40;i++) state.chainv[i] = c_IV[i];

		Update512(&state, Hash);
        finalization512(&state, Hash);
		//Cubehash

		uint32_t x[2][2][2][2][2];
		Init(x);
		// erste Hälfte des Hashes (32 bytes)
		Update32(x, Hash);
		// zweite Hälfte des Hashes (32 bytes)
		Update32(x, &Hash[8]);
		// Padding Block
		uint32_t last[8];
		last[0] = 0x80;
#pragma unroll 7
		for (int i = 1; i < 8; i++) last[i] = 0;
		Update32(x, last);
		Final(x, Hash);	
	}
}

__host__ void x11_luffaCubehash512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
    const uint32_t threadsperblock = 256;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

	x11_luffaCubehash512_gpu_hash_64 << <grid, block>> >(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);
}

