#include "hip/hip_runtime.h"
extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_keccak.h"
#include "lyra2/Lyra2.h"
}

#include "miner.h"
#include "cuda_helper.h"

static _ALIGN(64) uint64_t *d_hash[MAX_GPUS];

extern void blake256_cpu_init(int thr_id, uint32_t threads);
extern void blake256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash, int order);
extern void blake256_cpu_setBlock_80(uint32_t *pdata);
extern void keccak256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);
extern void keccak256_cpu_init(int thr_id, uint32_t threads);
extern void skein256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);
extern void skein256_cpu_init(int thr_id, uint32_t threads);

extern void lyra2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);
extern void lyra2_cpu_init(int thr_id, uint32_t threads);

extern void groestl256_setTarget(const void *ptarget);
extern void groestl256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_outputHash, int order, uint32_t *resultnonces);
extern void groestl256_cpu_init(int thr_id, uint32_t threads);

extern "C" void lyra2_hash(void *state, const void *input)
{
	sph_blake256_context     ctx_blake;
	sph_keccak256_context    ctx_keccak;
	sph_skein256_context     ctx_skein;
	sph_groestl256_context   ctx_groestl;

	uint32_t hashA[8], hashB[8];

	sph_blake256_init(&ctx_blake);
	sph_blake256(&ctx_blake, input, 80);
	sph_blake256_close(&ctx_blake, hashA);

	sph_keccak256_init(&ctx_keccak);
	sph_keccak256(&ctx_keccak, hashA, 32);
	sph_keccak256_close(&ctx_keccak, hashB);

	LYRA2(hashA, 32, hashB, 32, hashB, 32, 1, 8, 8);

	sph_skein256_init(&ctx_skein);
	sph_skein256(&ctx_skein, hashA, 32);
	sph_skein256_close(&ctx_skein, hashB);

	sph_groestl256_init(&ctx_groestl);
	sph_groestl256(&ctx_groestl, hashB, 32);
	sph_groestl256_close(&ctx_groestl, hashA);

	memcpy(state, hashA, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_lyra2(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	unsigned int intensity = (device_sm[device_map[thr_id]] > 500) ? 256 * 256 * 25 : 256 * 256 * 14;
	uint32_t throughput = device_intensity(thr_id, __func__, intensity); // 18=256*256*4;
	throughput = min(throughput, (max_nonce - first_nonce));

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x000f;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		hipDeviceReset();
		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		hipDeviceSetCacheConfig(hipFuncCachePreferL1);

		blake256_cpu_init(thr_id, throughput);
		keccak256_cpu_init(thr_id,throughput);
		skein256_cpu_init(thr_id, throughput);
		groestl256_cpu_init(thr_id, throughput);
		lyra2_cpu_init(thr_id, throughput);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput));

		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);

	blake256_cpu_setBlock_80(pdata);
	groestl256_setTarget(ptarget);

	do {
		int order = 0;
		uint32_t foundNonce[2] = { 0, 0 };

		blake256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		keccak256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		lyra2_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		skein256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
//		MyStreamSynchronize(NULL, 2, thr_id);
		groestl256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++, foundNonce);
		if (foundNonce[0] != 0)
		{
			CUDA_SAFE_CALL(hipGetLastError());
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce[0]);
			lyra2_hash(vhash64, endiandata);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				// check if there was some other ones...
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (foundNonce[1] != 0)
				{
					pdata[21] = foundNonce[1];
					res++;
					if (opt_benchmark)  applog(LOG_INFO, "GPU #%d Found second nounce %08x", thr_id, foundNonce[1], vhash64[7], Htarg);
				}
				pdata[19] = foundNonce[0];
				if (opt_benchmark) applog(LOG_INFO, "GPU #%d Found nounce % 08x", thr_id, foundNonce[0], vhash64[7], Htarg);
				return res;
			}
			else
			{
				if (vhash64[7] > Htarg) // don't show message if it is equal but fails fulltest
					applog(LOG_WARNING, "GPU #%d: result does not validate on CPU!", thr_id);
			}
		}

		pdata[19] += throughput;
	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
