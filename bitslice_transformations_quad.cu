
#include <hip/hip_runtime.h>
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 300
/**
 * __shfl() returns the value of var held by the thread whose ID is given by srcLane.
 * If srcLane is outside the range 0..width-1, the thread's own value of var is returned.
 */
#undef __shfl
#define __shfl(var, srcLane, width) (uint32_t)(var)
#endif

#define merge8(z,x,y)\
		z=__byte_perm(x, y, 0x5140); \

#define SWAP8(x,y)\
		x=__byte_perm(x, y, 0x5410); \
		y=__byte_perm(x, y, 0x7632);

#define SWAP4(x,y)\
		t = (y<<4); \
		t = (x ^ t); \
		t = 0xf0f0f0f0UL & t; \
		x = (x ^ t); \
		t=  t>>4;\
		y=  y ^ t;

#define SWAP2(x,y)\
		t = (y<<2); \
		t = (x ^ t); \
		t = 0xccccccccUL & t; \
		x = (x ^ t); \
		t=  t>>2;\
		y=  y ^ t;

#define SWAP1(x,y)\
		t = (y+y); \
		t = (x ^ t); \
		t = 0xaaaaaaaaUL & t; \
		x = (x ^ t); \
		t=  t>>1;\
		y=  y ^ t;


__device__ __forceinline__
void to_bitslice_quad(uint32_t *const __restrict__ input, uint32_t *const __restrict__ output)
{
    uint32_t other[8];
	uint32_t t;

    #pragma unroll
    for (int i = 0; i < 8; i++) 
	{
		const unsigned int n = threadIdx.x & 3;
		input[i] = __shfl((int)input[i], n ^ (3 * (n >= 1 && n <= 2)), 4);
        other[i] = __shfl((int)input[i], (threadIdx.x + 1) & 3, 4);
        input[i] = __shfl((int)input[i], threadIdx.x & 2, 4);
        other[i] = __shfl((int)other[i], threadIdx.x & 2, 4);
        if (threadIdx.x & 1) {
            input[i] = __byte_perm(input[i], 0, 0x1032);
            other[i] = __byte_perm(other[i], 0, 0x1032);
        }
    }

	merge8(output[0], input[0], input[4]);
	merge8(output[1], other[0], other[4]);
	merge8(output[2], input[1], input[5]);
	merge8(output[3], other[1], other[5]);
	merge8(output[4], input[2], input[6]);
	merge8(output[5], other[2], other[6]);
	merge8(output[6], input[3], input[7]);
	merge8(output[7], other[3], other[7]);

	SWAP1(output[0], output[1]);
	SWAP1(output[2], output[3]);
	SWAP1(output[4], output[5]);
	SWAP1(output[6], output[7]);

	SWAP2(output[0], output[2]);
	SWAP2(output[1], output[3]);
	SWAP2(output[4], output[6]);
	SWAP2(output[5], output[7]);

	SWAP4(output[0], output[4]);
	SWAP4(output[1], output[5]);
	SWAP4(output[2], output[6]);
	SWAP4(output[3], output[7]);
}

__device__ __forceinline__
void from_bitslice_quad(const uint32_t *const __restrict__ input, uint32_t *const __restrict__ output)
{

	uint32_t t;

	output[0] = __byte_perm(input[0], input[4], 0x7531);
	output[2] = __byte_perm(input[1], input[5], 0x7531);
	output[8] = __byte_perm(input[2], input[6], 0x7531);
	output[10] = __byte_perm(input[3], input[7], 0x7531);

	SWAP1(output[0], output[2]);
	SWAP1(output[8], output[10]);

	SWAP2(output[0], output[8]);
	SWAP2(output[2], output[10]);

	output[4] = __byte_perm(output[0], output[8], 0x5410);
	output[8] = __byte_perm(output[0], output[8], 0x7632);
	output[0] = output[4];

	output[6] = __byte_perm(output[2], output[10], 0x5410);
	output[10] = __byte_perm(output[2], output[10], 0x7632);
	output[2] = output[6];

	SWAP4(output[0], output[8]);
	SWAP4(output[2], output[10]);

	output[4] = output[0];
	output[6] = output[2];
	output[12] = output[8];
	output[14] = output[10];

	if (threadIdx.x & 1) 
	{
		output[0] = __byte_perm(output[0], 0, 0x1032);
		output[2] = __byte_perm(output[2], 0, 0x1032);
		output[4] = __byte_perm(output[4], 0, 0x3232);
		output[6] = __byte_perm(output[6], 0, 0x3232);
		output[8] = __byte_perm(output[8], 0, 0x1032);
		output[10] = __byte_perm(output[10], 0, 0x1032);
		output[12] = __byte_perm(output[12], 0, 0x3232);
		output[14] = __byte_perm(output[14], 0, 0x3232);
	}

	output[0] = __byte_perm(output[0], __shfl((int)output[0], (threadIdx.x + 1) & 3, 4), 0x7610);
	output[0 + 1] = __shfl((int)output[0], (threadIdx.x + 2) & 3, 4);

	output[2] = __byte_perm(output[2], __shfl((int)output[2], (threadIdx.x + 1) & 3, 4), 0x7610);
	output[2 + 1] = __shfl((int)output[2], (threadIdx.x + 2) & 3, 4);

	output[4] = __byte_perm(output[4], __shfl((int)output[4], (threadIdx.x + 1) & 3, 4), 0x7632);
	output[4 + 1] = __shfl((int)output[4], (threadIdx.x + 2) & 3, 4);

	output[6] = __byte_perm(output[6], __shfl((int)output[6], (threadIdx.x + 1) & 3, 4), 0x7632);
	output[6 + 1] = __shfl((int)output[6], (threadIdx.x + 2) & 3, 4);

	output[8] = __byte_perm(output[8], __shfl((int)output[8], (threadIdx.x + 1) & 3, 4), 0x7610);
	output[8 + 1] = __shfl((int)output[8], (threadIdx.x + 2) & 3, 4);

	output[10] = __byte_perm(output[10], __shfl((int)output[10], (threadIdx.x + 1) & 3, 4), 0x7610);
	output[10 + 1] = __shfl((int)output[10], (threadIdx.x + 2) & 3, 4);

	output[12] = __byte_perm(output[12], __shfl((int)output[12], (threadIdx.x + 1) & 3, 4), 0x7632);
	output[12 + 1] = __shfl((int)output[12], (threadIdx.x + 2) & 3, 4);

	output[14] = __byte_perm(output[14], __shfl((int)output[14], (threadIdx.x + 1) & 3, 4), 0x7632);
	output[14 + 1] = __shfl((int)output[14], (threadIdx.x + 2) & 3, 4);

/*	if (threadIdx.x & 3)
	{
		output[0] = output[0 + 1] = 0;
		output[2] = output[2 + 1] = 0;
		output[4] = output[4 + 1] = 0;
		output[6] = output[6 + 1] = 0;
		output[8] = output[8 + 1] = 0;
		output[10] = output[10 + 1] = 0;
		output[12] = output[12 + 1] = 0;
		output[14] = output[14 + 1] = 0;
	}
*/
}
